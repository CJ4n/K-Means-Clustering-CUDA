#include "hip/hip_runtime.h"
#include "kMeansGpu.h"
#include "findClosestCentriods.h"
#include "hip/hip_runtime.h"
#include "cudaCheckError.h"
#include "timer.h"
#include "Constants.h"

#define INDEX(f, c, tid, num_clusters, num_features) ((f * num_clusters) + c) + tid *(num_features + 1) * num_clusters
#define INDEX_ID(c, tid, num_clusters, num_features) ((num_features * num_clusters) + c) + tid *(num_features + 1) * num_clusters

// template <int NUM_FEATURES=2,int NUM_DATA_POINTS=200>
// __global__ void ReduceDataPoints(const DataPoints *points, const int k /*number of centroids*/, DataPoints *out, const int count_in, long *count_out, const int num_data_points)
__global__ void ReduceDataPoints(MyDataType **features, int *cluster_ids, MyDataType **centroids_features,
								 const int count_in, long *count_out, const int num_data_points, const int num_features, const int num_clusters)
{
	extern __shared__ MyDataType shm[];
	const int tid = threadIdx.x;
	const int gid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	// jakbybyły tempalte to można by trochę obliczeń zrobić w czasie kompilacji głownie indexy
	if (gid >= num_data_points)
	{
		return;
	}
	// // shm[(f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5}), (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5}),..., (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5}) ]
	// shm[(f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5},{count1,...,count5}), (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5},{count1,...,count5}),..., (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5},{count1,...,count5}) ]

	int c1, c2;

	for (int f = 0; f < num_features; ++f)
	{
		for (int c = 0; c < num_clusters; ++c)
		{
			shm[INDEX(f, c, tid, num_clusters, num_features)] = 0;
			shm[INDEX_ID(c, tid, num_clusters, num_features)] = 0;
		}

		c1 = cluster_ids[gid];
		shm[INDEX(f, c1, tid, num_clusters, num_features)] += features[f][gid];

		// problem: if datatype is long double then c ==-1 because in genertepoint  i set its clusterid to -1, but why onyl when long double?? ok its because shared memory is long double i gesss

		if (gid + blockDim.x >= num_data_points)
		{
			continue;
		}

		c2 = cluster_ids[gid + blockDim.x];
		shm[INDEX(f, c2, tid, num_clusters, num_features)] += features[f][gid + blockDim.x];
		// idx where to store particualr feature coord
	}
	{
		// int c = cluster_ids[gid];
		if (count_in)
			shm[INDEX_ID(c1, tid, num_clusters, num_features)] = count_in;
		else
			shm[INDEX_ID(c1, tid, num_clusters, num_features)] = count_out[gid];

		if (gid + blockDim.x < num_data_points)
		{
			if (count_in)
				shm[INDEX_ID(c2, tid, num_clusters, num_features)] += count_in;
			else
				shm[INDEX_ID(c2, tid, num_clusters, num_features)] += count_out[gid + blockDim.x];
		}
	}
	// else
	// 	return;
	__syncthreads();

	for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2)
	{
		if (tid < stride)
		{
			for (int f = 0; f < num_features; ++f)
			{
				for (int c = 0; c < num_clusters; ++c)
				{
					// czy to jest optumalny odczyt??
					shm[INDEX(f, c, tid, num_clusters, num_features)] += shm[INDEX(f, c, (tid + stride), num_clusters, num_features)];
					if (f == 0)
					{
						shm[INDEX_ID(c, tid, num_clusters, num_features)] += shm[INDEX_ID(c, (tid + stride), num_clusters, num_features)];
					}
				}
			}
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		for (int f = 0; f < num_features; ++f)
			for (int c = 0; c < num_clusters; ++c)
			{
				{
					centroids_features[f][c + blockIdx.x * num_clusters] = shm[INDEX(f, c, 0, num_clusters, num_features)];
					if (f == 0)
					{
						// [{count1,...,count5},{count1,...,count5},..,
						count_out[blockIdx.x * num_clusters + c] = shm[INDEX_ID(c, 0, num_clusters, num_features)];
					}
				}
			}
	}
}

#define INDEX_C(c, tid, num_clusters) c + tid *num_clusters
__global__ void ReduceDataPointsCountPoints(int *cluster_ids,
											const int count_in, long *count_out, const int num_data_points, const int num_features, const int num_clusters)
{
	extern __shared__ MyDataType shm[];
	const int tid = threadIdx.x;
	const int gid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	if (gid >= num_data_points)
	{
		return;
	}

	for (int c = 0; c < num_clusters; ++c)
	{
		shm[INDEX_C(c, tid, num_clusters)] = 0;
	}

	int c1 = cluster_ids[gid];

	if (count_in)
		shm[INDEX_C(c1, tid, num_clusters)] = count_in;
	else
		shm[INDEX_C(c1, tid, num_clusters)] = count_out[gid];

	if (gid + blockDim.x < num_data_points)
	{
		int c2 = cluster_ids[gid + blockDim.x];
		if (count_in)
			shm[INDEX_C(c2, tid, num_clusters)] += count_in;
		else
			shm[INDEX_C(c2, tid, num_clusters)] += count_out[gid + blockDim.x];
	}
	__syncthreads();

	for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2)
	{
		if (tid < stride)
		{
			for (int c = 0; c < num_clusters; ++c)
			{
				shm[INDEX_C(c, tid, num_clusters)] += shm[INDEX_C(c, tid, num_clusters)];
			}
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		for (int c = 0; c < num_clusters; ++c)
		{
			count_out[blockIdx.x * num_clusters + c] = shm[INDEX_C(c, tid, num_clusters)];
		}
	}
}

#define INDEX_F(c, tid, num_clusters) c + tid *num_clusters

__global__ void ReduceDataPointsByFeatures(MyDataType *features, int *cluster_ids, MyDataType *out,
										   const int num_data_points, const int num_clusters)
{
	extern __shared__ MyDataType shm[];
	const int tid = threadIdx.x;
	const int gid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	if (gid >= num_data_points)
	{
		return;
	}

	for (int c = 0; c < num_clusters; ++c)
	{
		shm[INDEX_F(c, tid, num_clusters)] = 0;
	}

	int c1 = cluster_ids[gid];
	shm[INDEX_F(c1, tid, num_clusters)] += features[gid];
	if (gid + blockDim.x >= num_data_points)
	{
		int c2 = cluster_ids[gid + blockDim.x];
		shm[INDEX_F(c2, tid, num_clusters)] += features[gid + blockDim.x];
	}
	__syncthreads();

	for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2)
	{
		if (tid < stride)
		{
			for (int c = 0; c < num_clusters; ++c)
			{
				shm[INDEX_F(c, tid, num_clusters)] += shm[INDEX_F(c, (tid + stride), num_clusters)];
			}
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		for (int c = 0; c < num_clusters; ++c)
		{
			out[c + blockIdx.x * num_clusters] = shm[INDEX_F(c, 0, num_clusters)];
		}
	}
}

__global__ void FindNewCentroids(DataPoints *centroids, long *count, DataPoints *reduced_points)
{
	// może zrobić to na talbicy wątkow dwuwymiarowej??
	const int gid = blockIdx.x * blockDim.x + threadIdx.x;
	const int f = gid / centroids->num_data_points;
	const int c = gid % centroids->num_data_points;
	if (gid >= centroids->num_features * centroids->num_data_points)
	{
		return;
	}
	centroids->features_array[f][c] = reduced_points->features_array[f][c] / (MyDataType)count[c];
}

__global__ void InitPointsWithCentroidsIds(DataPoints *points, int k, int num_points)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	if (gid >= num_points)
	{
		return;
	}
	points->cluster_id_of_point[gid] = gid % k;
}

#define DEBUG 0
#define MAX_SHM_SIZE 48 * 1024
#define DEFAULT_NUM_THREADS 1024l
#define CALCULATE_SHM_SIZE(num_features, num_clusters, num_threads) num_threads *(num_features + 1) * num_clusters * sizeof(MyDataType)

DataPoints *reduced_points;
long *ids_count;
int cur_epoch = 0;

void ReduceFeature(int w, DataPoints *feature, int num_features, int num_clusters, int num_datapoints, int count_in, long *count_out, int num_th, int num_bl)
{
	int num_threads = 1024;
	int num_blocks = (int)std::ceil(num_datapoints / (float)num_threads / 2.0);
	size_t shm_size = sizeof(MyDataType) * num_threads * num_clusters;
	for (int f = 0; f < num_features; ++f)
	{
		ReduceDataPointsByFeatures<<<num_blocks, num_threads, shm_size>>>(feature->features_array[f], feature->cluster_id_of_point, reduced_points->features_array[f], num_datapoints, num_clusters);
		hipDeviceSynchronize();
	}
	shm_size = sizeof(int) * num_threads * num_clusters;
	ReduceDataPointsCountPoints<<<num_blocks, num_threads, shm_size>>>(feature->cluster_id_of_point, count_in, count_out, num_datapoints, num_features, num_clusters);

	// size_t shm_size = CALCULATE_SHM_SIZE(num_features,num_clusters,num_th);
	// ReduceDataPoints<<<num_bl,num_th,shm_size>>>(feature->features_array,feature->cluster_id_of_point,reduced_points->features_array,count_in,count_out,num_datapoints,num_features,num_clusters);
	// hipDeviceSynchronize();
}

void debugFunction(DataPoints *points, DataPoints *debug, int num_features, int num_clusters, int num_blocks, int num_threads, int N, std::string label)
{
	std::cout << "\n---------" << label << "---------" << std::endl;
	long sum_tot = 0;
	// Gets exact sum by feature and clusters
	for (int f = 0; f < num_features; f++)
		for (int c = 0; c < num_clusters; c++)
		{
			debug->features_array[f][c] = 0;
		}

	for (int i = 0; i < points->num_data_points; i++)
	{
		for (int f = 0; f < num_features; f++)
		{
			debug->features_array[f][points->cluster_id_of_point[i]] += points->features_array[f][i];
			sum_tot += points->features_array[f][i];
		}
	}

	std::cout << " correct (points)\n{\n	";
	double sum_tot_v2 = 0;
	for (int c = 0; c < num_clusters; c++)
	{
		for (int f = 0; f < num_features; f++)
		{
			std::cout << debug->features_array[f][c] << ", ";
			sum_tot_v2 += debug->features_array[f][c];
			debug->features_array[f][c] = 0;
		}
	}
	std::cout << "\n}\n";
	// Gets exact sum by feature and clusters

	// Gets redcued sum by feature and cluster
	long sum_tot_reduced = 0;

	for (int i = 0; i < N; i++)
	{
		for (int f = 0; f < num_features; f++)
		{
			debug->features_array[f][reduced_points->cluster_id_of_point[i]] += reduced_points->features_array[f][i];
			sum_tot_reduced += reduced_points->features_array[f][i];
		}
	}
	std::cout << "Calculated points (reduced_points)\n{\n	";

	double sum_tot_reduced_v2 = 0;
	for (int c = 0; c < num_clusters; c++)
	{
		for (int f = 0; f < num_features; f++)
		{
			std::cout << debug->features_array[f][c] << ", ";
			sum_tot_reduced_v2 += debug->features_array[f][c];
			debug->features_array[f][c] = 0;
		}
	}
	std::cout << "\n}\n";

	// Gets redcued sum by feature and cluster

	std::cout << "sumed all points(sum_tot):           " << sum_tot << std::endl;
	std::cout << "sumed all points(sum_tot_v2)         " << sum_tot_v2 << std::endl;
	std::cout << "sumed all points(sum_tot_reduced)    " << sum_tot_reduced << std::endl;
	std::cout << "sumed all points(sum_tot_reduced_v2) " << sum_tot_reduced_v2 << std::endl;

	int *count_check = (int *)malloc(sizeof(int) * num_clusters);
	memset(count_check, 0, sizeof(int) * num_clusters);

	// Gets exact count of ids
	long exact_points_count = 0;
	for (int i = 0; i < points->num_data_points; ++i)
	{
		count_check[points->cluster_id_of_point[i]]++;
	}
	std::cout << "Exact ids count\n{\n	";
	for (int c = 0; c < num_clusters; ++c)
	{
		std::cout << count_check[c] << ", ";
		exact_points_count += count_check[c];
	}
	std::cout << "\n}\n";
	// Gets exact count of ids

	memset(count_check, 0, sizeof(int) * num_clusters);

	// Gets reduced count of ids
	long reduced_points_count = 0;
	for (int i = 0; i < num_blocks; ++i)
	{
		for (int c = 0; c < num_clusters; ++c)
		{
			count_check[c] += ids_count[i * num_clusters + c];
			reduced_points_count += ids_count[i * num_clusters + c];
		}
	}
	std::cout << "Reduced ids count\n{\n	";
	for (int c = 0; c < num_clusters; ++c)
	{
		std::cout << count_check[c] << ", ";
	}
	std::cout << "\n}\n";
	// Gets reduced count of ids

	std::cout << "number of points (exact_points_count):   " << exact_points_count << std::endl;
	std::cout << "number of points (reduced_points_count): " << reduced_points_count << std::endl;

	free(count_check);
	if (num_blocks != -1)
		if (num_blocks * num_threads * 2 < N || N != num_clusters * num_blocks)
		{
			std::cout << "aaaaaaaaaaaaaaaaaaaaaa\n";
		}
}

void KMeansOneIterationGpu(DataPoints *points, DataPoints *centroids)
{
	const int num_features = points->num_features;
	const int num_clusters = centroids->num_data_points;
	int N = points->num_data_points;
	int num_threads = DEFAULT_NUM_THREADS;

	while (MAX_SHM_SIZE < CALCULATE_SHM_SIZE(num_features, num_clusters, num_threads))
	{
		num_threads /= 2;
	}

	long num_blocks = (int)std::max(std::ceil((long)(N / (double)num_threads)), 1.0);
	size_t shmem_size = CALCULATE_SHM_SIZE(num_features, num_clusters, num_threads);

	DataPoints *debug;
	if (DEBUG)
	{
		debug = AllocateDataPoints(num_features, num_clusters);
	}

	// Find closest centroids for each datapoint
	const int num_threads_find_closest = 1024;
	const int num_blocks_find_closest = std::max(1, (int)std::ceil(points->num_data_points / num_threads_find_closest));
	const size_t shm_find_closest = sizeof(MyDataType) * num_clusters * num_features + sizeof(MyDataType) * num_threads_find_closest * num_features;
	timer_find_closest_centroids->Start();
	FindClosestCentroids<<<num_blocks_find_closest, num_threads_find_closest, shm_find_closest>>>(points->features_array,
																								  points->cluster_id_of_point, centroids->features_array, points->num_data_points,
																								  num_features, num_clusters);
	timer_find_closest_centroids->Stop();
	timer_find_closest_centroids->Elapsed();
	cudaCheckError();
	// Find closest centroids for each datapoint

	auto lambda = [](int n)
	{
		unsigned count = 0;
		if (n && !(n & (n - 1)))
			return n;

		while (n != 0)
		{
			n >>= 1;
			count += 1;
		}

		return 1 << count;
	};

	// Create and init reduced points, what will be used sum up all points
	num_blocks = std::ceil(num_blocks / 2.0);
	// tmp = lambda(tmp);
	long num_reduced_points = num_blocks * num_clusters;
	if (cur_epoch == 0)
	{
		reduced_points = AllocateDataPoints(num_features, num_reduced_points);
		int num_threads_inti_id = (int)std::min(DEFAULT_NUM_THREADS, num_reduced_points);
		int num_block_init_id = (int)std::max(std::ceil((num_reduced_points / (double)num_threads_inti_id)), 1.0);
		InitPointsWithCentroidsIds<<<num_block_init_id, num_threads_inti_id>>>(reduced_points, num_clusters, num_reduced_points);
		hipDeviceSynchronize();
		cudaCheckError();
		hipMallocManaged(&ids_count, sizeof(long) * num_blocks * num_clusters);
		cudaCheckError();
		hipMemset(ids_count, 0, sizeof(long) * num_blocks * num_clusters);
		cudaCheckError();
	}
	else
	{
		// clean recued_points??
	}
	// Create and init reduced points

	if (DEBUG)
	{
		debugFunction(points, debug, num_features, num_clusters, num_blocks, num_threads, num_clusters * num_blocks, "BEFORE FIRST REDUCE");
	}
	// reduce points in `points` and store them in `reduced_poitsn`
	timer_compute_centroids->Start();
	ReduceDataPoints<<<num_blocks, num_threads, shmem_size>>>(points->features_array,
															  points->cluster_id_of_point, reduced_points->features_array,
															  1, ids_count, N, num_features, num_clusters);
	// ReduceFeature(0, points, num_features, num_clusters, N, 1, ids_count, num_blocks, num_threads);

	timer_compute_centroids->Stop();
	timer_compute_centroids->Elapsed();
	cudaCheckError();
	// reduce points in `points` and store them in reduced_poitsn

	if (DEBUG)
	{
		debugFunction(points, debug, num_features, num_clusters, num_blocks, num_threads, num_clusters * num_blocks, "BEFORE WHILE REDUCE");
	}
	// further reduce points in `reduced_points`, until there will be no more then  `num_threads * 2` poitns left to reduce
	while (num_blocks * num_clusters > num_threads * 2)
	{
		N = num_blocks * num_clusters;
		// N=lambda(N);
		num_blocks = std::ceil(N / num_threads / 2.0);
		shmem_size = CALCULATE_SHM_SIZE(num_features, num_clusters, num_threads);

		timer_compute_centroids->Start();
		ReduceDataPoints<<<num_blocks, num_threads, shmem_size>>>(reduced_points->features_array,
																  reduced_points->cluster_id_of_point, reduced_points->features_array,
																  0, ids_count, N, num_features, num_clusters);
		timer_compute_centroids->Stop();
		timer_compute_centroids->Elapsed();

		cudaCheckError();
	}
	if (DEBUG)
	{
		debugFunction(points, debug, num_features, num_clusters, num_blocks, num_threads, num_blocks * num_clusters, "AFTER WHILE REDUCE");
	}
	// further reduce points in `reduced_points`, until there will be no more then  `num_threads * 2` poitns left to reduce

	// last reduce, reduce all remaining points to a 'single datapoint', that is: points belonging to the same cluster will be reduced to single point
	if (num_blocks > 1) // if may happen, that last reduced reduced all the point, happen when num_blocks==1
	{
		N = num_clusters * num_blocks;
		int num_threads_last_sumup = std::ceil(N / 2.0);
		num_threads_last_sumup = lambda(num_threads_last_sumup);

		shmem_size = CALCULATE_SHM_SIZE(num_features, num_clusters, num_threads_last_sumup);
		timer_compute_centroids->Start();
		ReduceDataPoints<<<1, num_threads_last_sumup, shmem_size>>>(reduced_points->features_array,
																	reduced_points->cluster_id_of_point, reduced_points->features_array,
																	0, ids_count, N, num_features, num_clusters);
		timer_compute_centroids->Stop();
		timer_compute_centroids->Elapsed();
		cudaCheckError();
		if (DEBUG)
		{
			debugFunction(points, debug, num_features, num_clusters, 1, num_threads_last_sumup, 1 * num_clusters, "AFTER LAST REDUCE");
		}
	}
	// last reduce, reduce all remaining points

	// find new centroids
	FindNewCentroids<<<1, num_features * num_clusters>>>(centroids, ids_count, reduced_points);
	hipDeviceSynchronize();
	cudaCheckError();
	// find new centroids

	// cleanup memory
	if (cur_epoch == constants::num_epoches - 1)
	{
		DeallocateDataPoints(reduced_points);
		hipFree(ids_count);
		cudaCheckError();
	}
	if (DEBUG)
	{
		DeallocateDataPoints(debug);
	}
	cudaCheckError();
}