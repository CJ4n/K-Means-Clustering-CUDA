#include "hip/hip_runtime.h"
#include "kMeansGpu.h"
#include "findClosestCentriods.h"
#include "hip/hip_runtime.h"
#include "cudaCheckError.h"
#include "timer.h"

#define INDEX(f, c, tid, feature_stide, k, num_features) ((f * feature_stirde) + c) + tid *(num_features + 1) * k
#define INDEX_ID(c, tid, feature_stide, k, num_features) ((num_features * feature_stirde) + c) + tid *(num_features + 1) * k

// template <int NUM_FEATURES=2,int NUM_DATA_POINTS=200>
__global__ void ReduceDataPoints(const DataPoints *points, int k /*number of centroids*/, DataPoints *out, int count_in, int *count_out, int num_data_points)
{
	extern __shared__ float shm[];
	int tid = threadIdx.x;
	int gid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	int feature_stirde = k;
	int num_clusters = k;
	int num_features = points->num_features;

	// // shm[(f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5}), (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5}),..., (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5}) ]
	// shm[(f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5},{count1,...,count5}), (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5}),..., (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5}) ]
	for (int f = 0; f < points->num_features; ++f)
	{

		for (int c = 0; c < k; ++c)
		{
			// shm[((f * feature_stirde) + c) + tid * points->num_features * k] = 0;
			shm[INDEX(f, c, tid, feature_stirde, k, num_features)] = 0;
			shm[INDEX_ID(c, tid, feature_stirde, k, num_features)] = 0;
		}
		if (gid + blockDim.x >= num_data_points)
		{
			break;;
		}

		int c = points->cluster_id_of_point[gid];

		// shm[((f * feature_stirde) + c) + tid * points->num_features * k] += points->features_array[f][gid];
		shm[INDEX(f, c, tid, feature_stirde, k, num_features)] = points->features_array[f][gid];

		c = points->cluster_id_of_point[gid + blockDim.x];
		// shm[((f * feature_stirde) + c) + tid * points->num_features * k] += points->features_array[f][gid + blockDim.x];
		shm[INDEX(f, c, tid, feature_stirde, k, num_features)] += points->features_array[f][gid + blockDim.x];
		// shm[INDEX_ID(c, tid, feature_stirde, k, points->num_features)] += 1;
		// idx where to store particualr feature coord
	}

	int c = points->cluster_id_of_point[gid];
	if (count_in)
		shm[INDEX_ID(c, tid, feature_stirde, k, num_features)] = count_in;
	else
		shm[INDEX_ID(c, tid, feature_stirde, k, num_features)] = count_out[gid];

	c = points->cluster_id_of_point[gid + blockDim.x];
	if (count_in)
		shm[INDEX_ID(c, tid, feature_stirde, k, num_features)] += count_in;
	else
		shm[INDEX_ID(c, tid, feature_stirde, k, num_features)] += count_out[gid + blockDim.x];

	__syncthreads();

	for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2)
	{
		// problem jest gdy num_threads nie jest wiekokrotnoscia 2, wtedy jak mamy blockdim.x/2 itd, dostanimy cos co sie nie podzieli przez 2
		if (tid < stride)
		{
			for (int f = 0; f < num_features; ++f)
			{
				for (int c = 0; c < k; ++c)
				{
					// czy to jest optumalny odczyt??
					// shm[((f * feature_stirde) + c) + tid * points->num_features * k] += shm[((f * feature_stirde) + c) + (tid + stride) * points->num_features * k];
					shm[INDEX(f, c, tid, feature_stirde, k, num_features)] += shm[INDEX(f, c, (tid + stride), feature_stirde, k, num_features)];
				}
			}
			for (int c = 0; c < k; ++c)
			{
				shm[INDEX_ID(c, tid, feature_stirde, k, num_features)] += shm[INDEX_ID(c, (tid + stride), feature_stirde, k, num_features)];
			}
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		for (int f = 0; f < num_features; ++f)
		{
			for (int c = 0; c < k; ++c)
			{
				// out->features_array[f][c + blockIdx.x * k] = shm[((f * feature_stirde) + c)];
				out->features_array[f][c + blockIdx.x * k] = shm[INDEX(f, c, 0, feature_stirde, k, num_features)];
			}
		}
		for (int c = 0; c < k; ++c)
		{
			// [{count1,...,count5},{count1,...,count5},..,
			count_out[blockIdx.x * k + c] = shm[INDEX_ID(c, 0, feature_stirde, k, num_features)];
		}
	}
}

__global__ void FindNewCentroids(DataPoints *centroids, int *count, DataPoints *reduced_points)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	int f = gid / centroids->num_data_points;
	int c = gid % centroids->num_data_points;
	centroids->features_array[f][c] = reduced_points->features_array[f][c] / (float)count[c];
}

__global__ void InitPointsWithCentroidsIds(DataPoints *points, int k)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	points->cluster_id_of_point[gid] = gid % k;
}

#include <iostream>
void KMeansOneIterationGpu(DataPoints *points, DataPoints *centroids)
{
	const int num_features = points->num_features;
	const int num_clusters = centroids->num_data_points;
	const int N = points->num_data_points;
	const int num_threads = 1024 / 4;
	int num_blocks = (int)std::max(std::ceil((int)(N / num_threads)), 1.0);
	size_t shmem_size = num_threads * sizeof(float) * num_features * num_clusters + num_threads * sizeof(float) * num_clusters;

	timer_find_closest_centroids.Start();
	FindClosestCentroids<<<num_blocks, num_threads>>>(points, centroids);
	timer_find_closest_centroids.Stop();
	timer_find_closest_centroids.Elapsed();
	cudaCheckError();

	num_blocks = std::ceil(num_blocks / 2);
	// rewerite lambda
	auto lambda = [](int n)
	{
		unsigned count = 0;
		if (n && !(n & (n - 1)))
			return n;

		while (n != 0)
		{
			n >>= 1;
			count += 1;
		}

		return 1 << count;
	};
	int tmp = num_blocks * num_clusters;
	tmp = lambda(tmp);
	DataPoints *out = AllocateDataPoints(num_features, tmp);
	const int num_threads_inti_id = std::min(1024, tmp);
	const int num_block_init_id = (int)std::max(std::ceil((int)(tmp / num_threads_inti_id)), 1.0);
	InitPointsWithCentroidsIds<<<num_block_init_id, num_threads_inti_id>>>(out, num_clusters);
	// for(int i=0;i<num_block_init_id*num_threads_inti_id;i++){
	// 	std::cout<<out->cluster_id_of_point[i]<<", ";
	// }
	// std::cout<<std::endl;
	hipDeviceSynchronize();
	cudaCheckError();

	int *count_out;
	hipMallocManaged(&count_out, sizeof(int) * num_blocks * num_clusters);
	cudaCheckError();
	hipMemset(count_out, 0, sizeof(int) * num_blocks * num_clusters);
	cudaCheckError();

	// second reduce
	timer_compute_centroids.Start();
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	if (num_blocks * num_threads * 2 != points->num_data_points)
	{
		std::cout << "11aaaaaaaaaaaaaaaaaaaaaa\n";
	}
	for (int i = 0; i < num_blocks * num_clusters; i++)
	{
		count_out[i] = 1;
	}
	ReduceDataPoints<<<num_blocks, num_threads, shmem_size>>>(points, num_clusters, out, 1, count_out, N);
	// jak gdyby to problem bo sa paski zer
	timer_compute_centroids.Stop();
	timer_compute_centroids.Elapsed();
	cudaCheckError();
	for (int i = 0; i < num_blocks * num_clusters; i++)
	{
		std::cout << count_out[i] << ", ";
	}
	std::cout << std::endl;
	// second reduce
	for (int i = 0; i < num_blocks * num_clusters; i++)
	{
		// std::cout<<count_out[i]<<",";
	}
	std::cout << std::endl;
	// num_threads = 1024 / 2;
	shmem_size = num_threads * sizeof(float) * num_features * num_clusters + num_threads * sizeof(float) * num_clusters;
	const int new_num_block = std::max(tmp / num_threads / 2, 1);
	// new_num_block = lambda(new_num_block);
	if (new_num_block * num_threads * 2 != out->num_data_points)
	{
		std::cout << "222aaaaaaaaaaaaaaaaaaaaaa\n";
	}

	ReduceDataPoints<<<new_num_block, num_threads, shmem_size>>>(out, num_clusters, out, 0, count_out, tmp);
	hipDeviceSynchronize();
	cudaCheckError();

	timer_compute_centroids.Start();
	const int num_threads_last_sumup = new_num_block / 2 * num_clusters;
	if (new_num_block > 1)
	{
		if (1 * 2 * num_threads_last_sumup != new_num_block * num_clusters)
		{
			std::cout << "333aaaaaaaaaaaaaaaaaaaaaa\n";
		}
		shmem_size = num_threads_last_sumup * sizeof(float) * num_features * num_clusters + num_threads_last_sumup * sizeof(float) * num_clusters;

		ReduceDataPoints<<<1, num_threads_last_sumup, shmem_size>>>(out, num_clusters, out, 0, count_out, tmp);
	}
	timer_compute_centroids.Stop();
	timer_compute_centroids.Elapsed();
	cudaCheckError();

	// FindNewCentroids<<<1, num_features * num_clusters>>>(centroids, count_out, out);
	hipDeviceSynchronize();
	cudaCheckError();

	for (int f = 0; f < num_features; f++)
		for (int c = 0; c < num_clusters; c++)
		{
			centroids->features_array[f][c] = out->features_array[f][c] / count_out[c];
			// 	std::cout<<centroids->features_array[f][c] <<", "<< out->features_array[f][c]<<", "<<count_out[c];
			// std::cout<<std::endl;
		}

	DeallocateDataPoints(out);
	hipFree(count_out);
	cudaCheckError();
}