#include "hip/hip_runtime.h"
#include "kMeansGpu.h"
#include "findClosestCentriods.h"
#include "hip/hip_runtime.h"
#include "cudaCheckError.h"
#include "timer.h"

#define INDEX(f, c, tid, feature_stide, k, num_features) ((f * feature_stirde) + c) + tid *(num_features + 1) * k
#define INDEX_ID(c, tid, feature_stide, k, num_features) ((num_features * feature_stirde) + c) + tid *(num_features + 1) * k

// template <int NUM_FEATURES=2,int NUM_DATA_POINTS=200>
__global__ void ReduceDataPoints(const DataPoints *points, int k /*number of centroids*/, DataPoints *out, int count_in, int *count_out, int num_data_points)
{
	extern __shared__ float shm[];
	int tid = threadIdx.x;
	int gid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	int feature_stirde = k;
	int num_clusters = k;
	int num_features = points->num_features;

	// // shm[(f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5}), (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5}),..., (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5}) ]
	// shm[(f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5},{count1,...,count5}), (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5}),..., (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5}) ]
	for (int f = 0; f < points->num_features; ++f)
	{

		for (int c = 0; c < num_clusters; ++c)
		{
			// shm[((f * feature_stirde) + c) + tid * points->num_features * k] = 0;
			shm[INDEX(f, c, tid, feature_stirde, k, num_features)] = 0;
			shm[INDEX_ID(c, tid, feature_stirde, k, num_features)] = 0;
		}

		int c = points->cluster_id_of_point[gid];

		// shm[((f * feature_stirde) + c) + tid * points->num_features * k] += points->features_array[f][gid];
		shm[INDEX(f, c, tid, feature_stirde, k, num_features)] = points->features_array[f][gid];
		if (gid + blockDim.x >= num_data_points)
		{
			continue;
		}

		c = points->cluster_id_of_point[gid + blockDim.x];
		// shm[((f * feature_stirde) + c) + tid * points->num_features * k] += points->features_array[f][gid + blockDim.x];
		shm[INDEX(f, c, tid, feature_stirde, k, num_features)] += points->features_array[f][gid + blockDim.x];
		// shm[INDEX_ID(c, tid, feature_stirde, k, points->num_features)] += 1;
		// idx where to store particualr feature coord
	}

	int c = points->cluster_id_of_point[gid];
	if (count_in)
		shm[INDEX_ID(c, tid, feature_stirde, k, num_features)] = count_in;
	else
		shm[INDEX_ID(c, tid, feature_stirde, k, num_features)] = count_out[gid];

	if (gid + blockDim.x < num_data_points)
	{
		c = points->cluster_id_of_point[gid + blockDim.x];
		if (count_in)
			shm[INDEX_ID(c, tid, feature_stirde, k, num_features)] += count_in;
		else
			shm[INDEX_ID(c, tid, feature_stirde, k, num_features)] += count_out[gid + blockDim.x];
	}
	else
		return;
	__syncthreads();

	for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2)
	{
		// problem jest gdy num_threads nie jest wiekokrotnoscia 2, wtedy jak mamy blockdim.x/2 itd, dostanimy cos co sie nie podzieli przez 2
		if (tid < stride)
		{
			for (int f = 0; f < num_features; ++f)
			{
				for (int c = 0; c < num_clusters; ++c)
				{
					// czy to jest optumalny odczyt??
					// shm[((f * feature_stirde) + c) + tid * points->num_features * k] += shm[((f * feature_stirde) + c) + (tid + stride) * points->num_features * k];
					shm[INDEX(f, c, tid, feature_stirde, k, num_features)] += shm[INDEX(f, c, (tid + stride), feature_stirde, k, num_features)];
				}
			}
			for (int c = 0; c < num_clusters; ++c)
			{
				shm[INDEX_ID(c, tid, feature_stirde, k, num_features)] += shm[INDEX_ID(c, (tid + stride), feature_stirde, k, num_features)];
			}
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		for (int f = 0; f < num_features; ++f)
		{
			for (int c = 0; c < num_clusters; ++c)
			{
				// out->features_array[f][c + blockIdx.x * k] = shm[((f * feature_stirde) + c)];
				out->features_array[f][c + blockIdx.x * k] = shm[INDEX(f, c, 0, feature_stirde, k, num_features)];
			}
		}
		for (int c = 0; c < num_clusters; ++c)
		{
			// [{count1,...,count5},{count1,...,count5},..,
			count_out[blockIdx.x * k + c] = shm[INDEX_ID(c, 0, feature_stirde, k, num_features)];
		}
	}
}

__global__ void FindNewCentroids(DataPoints *centroids, int *count, DataPoints *reduced_points)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	int f = gid / centroids->num_data_points;
	int c = gid % centroids->num_data_points;
	centroids->features_array[f][c] = reduced_points->features_array[f][c] / (float)count[c];
}

__global__ void InitPointsWithCentroidsIds(DataPoints *points, int k, int num_points)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	if (gid >= num_points)
	{
		return;
	}
	points->cluster_id_of_point[gid] = gid % k;
}

#include <iostream>
void KMeansOneIterationGpu(DataPoints *points, DataPoints *centroids)
{
	const int num_features = points->num_features;
	const int num_clusters = centroids->num_data_points;
	int N = points->num_data_points;
	const int num_threads = 1024 / 4;
	int num_blocks = (int)std::max(std::ceil((int)(N / (double)num_threads)), 1.0);
	size_t shmem_size = num_threads * sizeof(float) * num_features * num_clusters + num_threads * sizeof(float) * num_clusters;

	timer_find_closest_centroids->Start();
	FindClosestCentroids<<<num_blocks, num_threads>>>(points, centroids);
	timer_find_closest_centroids->Stop();
	timer_find_closest_centroids->Elapsed();
	cudaCheckError();
	int sum_tot = 0;
	if (0)
	{
		int *exact_num = new int[num_clusters];
		for (int i = 0; i < num_clusters; i++)
		{
			exact_num[i] = 0;
		}
		for (int i = 0; i < N; i++)
		{
			exact_num[points->cluster_id_of_point[i]]++;
		}
		std::cout << std::endl;
		int sum_tot = 0;
		for (int i = 0; i < num_clusters; i++)
		{
			sum_tot += exact_num[i];
			std::cout << exact_num[i] << ", ";
		}
		free(exact_num);
		std::cout << std::endl
				  << N << " total sum: " << sum_tot << std::endl;
	}

	num_blocks = std::ceil(num_blocks / 2.0);
	// rewerite lambda
	auto lambda = [](int n)
	{
		unsigned count = 0;
		if (n && !(n & (n - 1)))
			return n;

		while (n != 0)
		{
			n >>= 1;
			count += 1;
		}

		return 1 << count;
	};
	int tmp = num_blocks * num_clusters;
	// tmp = lambda(tmp);
	DataPoints *out = AllocateDataPoints(num_features, tmp);
	const int num_threads_inti_id = std::min(1024, tmp);
	const int num_block_init_id = (int)std::max(std::ceil((int)(tmp / (double)num_threads_inti_id)), 1.0);
	InitPointsWithCentroidsIds<<<num_block_init_id, num_threads_inti_id>>>(out, num_clusters, tmp);
	hipDeviceSynchronize();
	cudaCheckError();

	int *count_out;
	hipMallocManaged(&count_out, sizeof(int) * num_blocks * num_clusters);
	cudaCheckError();
	hipMemset(count_out, 0, sizeof(int) * num_blocks * num_clusters);
	cudaCheckError();

	// for(int i=0;i<num_blocks*num_clusters;i++){
	// 	count_out[i]=1;
	// }
	// second reduce
	timer_compute_centroids->Start();
	// hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	if (num_blocks * num_threads * 2 != points->num_data_points || out->num_data_points != num_clusters * num_blocks)
	{
		std::cout << "11aaaaaaaaaaaaaaaaaaaaaa\n";
	}
	// }
	ReduceDataPoints<<<num_blocks, num_threads, shmem_size>>>(points, num_clusters, out, 1, count_out, N);
	// jak gdyby to problem bo sa paski zer
	timer_compute_centroids->Stop();
	timer_compute_centroids->Elapsed();
	cudaCheckError();
	sum_tot = 0;
	for (int i = 0; i < num_clusters * num_blocks; i++)
	{
		sum_tot += count_out[i];
		// std::cout << "coutout: " << count_out[i] << ",  ";
	}
	// std::cout << "\n tot_sum" << sum_tot << "N: " << points->num_data_points << std::endl;
	// for (int i = 0; i < tmp; i++)
	// {
	// 	// for (int c = 0; c < num_clusters; c++)
	// 		for (int f = 0; f < num_features; f++)
	// 	{
	// 		{
	// 			std::cout << out->features_array[f][i] << ", ";
	// 		}
	// 	}
	// 	if ((i % num_clusters) == 0)
	// 		std::cout << std::endl;
	// }

	// std::cout << std::endl;
	N = num_blocks * num_clusters;
	// N=lambda(N);
	const int new_num_block = std::ceil(N / num_threads / 2.0);
	shmem_size = num_threads * sizeof(float) * num_features * num_clusters + num_threads * sizeof(float) * num_clusters;
	if (new_num_block * num_threads * 2 != tmp)
	{
		std::cout << "222aaaaaaaaaaaaaaaaaaaaaa\n";
	}

	// DataPoints *out_new = AllocateDataPoints(num_features, N);
	// int out_num_threads = (N < 1024) ? N : 1024;
	// int out_num_blocks = (N < 1024) ? 1 : std::ceil(N / out_num_threads);
	// InitPointsWithCentroidsIds<<<out_num_blocks, out_num_threads>>>(out_new, num_clusters, N);
	// hipDeviceSynchronize();
	// cudaCheckError();
	// N = 2 * new_num_block * num_threads;
	ReduceDataPoints<<<new_num_block, num_threads, shmem_size>>>(out, num_clusters, out, 0, count_out, N);
	hipDeviceSynchronize();
	cudaCheckError();
	// sum_tot = 0;
	// for (int i = 0; i < num_clusters * new_num_block; i++)
	// {
	// 	sum_tot += count_out[i];
	// 	// std::cout << "coutout: " << count_out[i] << ",  ";
	// }
	// std::cout << "\n ||tot_sum" << sum_tot << "N: " << points->num_data_points << std::endl;
	timer_compute_centroids->Start();
	N = num_clusters * new_num_block;
	int num_threads_last_sumup = std::ceil(N / 2.0);
	num_threads_last_sumup = lambda(num_threads_last_sumup);
	if (new_num_block > 1)
	{
		if (1 * 2 * num_threads_last_sumup < N)
		{
			std::cout << "333aaaaaaaaaaaaaaaaaaaaaa\n";
		}
		shmem_size = num_threads_last_sumup * sizeof(float) * num_features * num_clusters + num_threads_last_sumup * sizeof(float) * num_clusters;
		// for(int i=num_threads_last_sumup*2;i<N;i++){
		// 	count_out[i]=0;
		// }
		// problem z nieparzystumi liczbammi
		ReduceDataPoints<<<1, num_threads_last_sumup, shmem_size>>>(out, num_clusters, out, 0, count_out, N);
	}
	timer_compute_centroids->Stop();
	timer_compute_centroids->Elapsed();
	cudaCheckError();

	FindNewCentroids<<<1, num_features * num_clusters>>>(centroids, count_out, out);
	hipDeviceSynchronize();
	cudaCheckError();
	// sum_tot = 0;
	// for (int i = 0; i < num_clusters * 1; i++)
	// {
	// 	sum_tot += count_out[i];
	// 	std::cout << "coutout: " <<sum_tot << ",  ";
	// }
	// std::cout << "\n tot_sum" << sum_tot << "N: " << points->num_data_points << std::endl;
	// for (int f = 0; f < num_features; f++)
	// 	for (int c = 0; c < num_clusters; c++)
	// 	{
	// 		centroids->features_array[f][c] = out_new->features_array[f][c] / count_out[c];
	// 		// std::cout<<centroids->features_array[f][c] <<", "<< out->features_array[f][c]<<", "<<count_out[c];
	// 		// std::cout<<std::endl;
	// 	}

	DeallocateDataPoints(out);
	// DeallocateDataPoints(out_new);
	hipFree(count_out);
	cudaCheckError();
}