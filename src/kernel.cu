#include "hip/hip_runtime.h"


#include <cstdlib>
#include <hip/hip_runtime.h>
#include <math.h>
#include <ctime>	// for a random seed
#include <fstream>	// for file-reading
#include <iostream> // for file-reading
#include <sstream>	// for file-reading
#include <vector>
#include <math.h>
// #include <thrust/host_vector.h>
#include <thrust/device_vector.h>
// #include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include "dataPoints.h"

#define cudaCheckError()                                                                    \
	{                                                                                       \
		hipError_t e = hipGetLastError();                                                 \
		if (e != hipSuccess)                                                               \
		{                                                                                   \
			printf("Cudafailure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
			exit(0);                                                                        \
		}                                                                                   \
	}

// struct Point
// {
// 	double x, y;	// coordinates
// 	int cluster;	// no default cluster
// 	double minDist; // default infinite dist to nearest cluster

// 	Point() : x(0.0),
// 			  y(0.0),
// 			  cluster(-1),
// 			  minDist(__DBL_MAX__) {}

// 	Point(double x, double y) : x(x),
// 								y(y),
// 								cluster(-1),
// 								minDist(__DBL_MAX__) {}

// 	double distance(Point p)
// 	{
// 		return (p.x - x) * (p.x - x) + (p.y - y) * (p.y - y);
// 	}
// };

// struct pt
// {
// 	double **features_array;
// 	int *cluster_id_of_point;
// 	double *minDist_to_cluster;
// 	int num_data_points;
// 	int num_features;
// };

// pt *allocate_pt(int num_features, int num_data_points)
// {

// 	pt *point;
// 	hipMallocManaged(&point, sizeof(pt));
// 	cudaCheckError();

// 	point->num_data_points = num_data_points;
// 	hipMallocManaged(&(point->cluster_id_of_point), sizeof(int) * num_data_points);
// 	cudaCheckError();
// 	hipMallocManaged(&(point->minDist_to_cluster), sizeof(double) * num_data_points);
// 	cudaCheckError();

// 	point->num_features = num_features;
// 	hipMallocManaged(&(point->features_array), sizeof(double *) * point->num_features);
// 	cudaCheckError();

// 	for (int feature = 0; feature < point->num_features; ++feature)
// 	{
// 		hipMallocManaged(&(point->features_array[feature]), sizeof(double) * point->num_data_points);
// 		cudaCheckError();
// 	}
// 	return point;
// }

// pt *readCsv()
// {
// 	std::vector<Point> points;
// 	std::string line;
// 	std::ifstream file("/home/jan/Desktop/K-Means-Clustering-CUDA/mall_data.csv");
// 	// std::ifstream file("../mall_data.csv");
// 	while (std::getline(file, line))
// 	{
// 		std::stringstream lineStream(line);
// 		std::string bit;
// 		double x, y;
// 		getline(lineStream, bit, ',');
// 		x = std::stof(bit);
// 		getline(lineStream, bit, '\n');
// 		y = stof(bit);

// 		points.push_back(Point(x, y));
// 	}
// 	file.close();

// 	pt *point = allocate_pt(2, points.size());
// 	int i = 0;
// 	for (std::vector<Point>::iterator it = points.begin(); it != points.end(); ++it)
// 	{
// 		double XY[2];
// 		XY[0] = it->x;
// 		XY[1] = it->y;
// 		for (int feature = 0; feature < point->num_features; ++feature)
// 		{
// 			point->features_array[feature][i] = XY[feature];
// 		}
// 		point->cluster_id_of_point[i] = it->cluster;
// 		point->minDist_to_cluster[i] = __DBL_MAX__;
// 		i++;
// 	}
// 	return point;
// }

// void saveCsv(pt *point, std::string file_name)
// {
// 	std::ofstream myfile;
// 	std::remove(file_name.c_str());
// 	myfile.open(file_name);
// 	myfile << "x,y,c" << std::endl;

// 	for (int i = 0; i < point->num_data_points; ++i)
// 	{
// 		for (int feature = 0; feature < point->num_features; ++feature)
// 		{
// 			myfile << point->features_array[feature][i];
// 			myfile << ",";
// 		}

// 		myfile << point->cluster_id_of_point[i] << std::endl;
// 	}
// 	myfile.close();
// }

double distance(dataPoints *p1, dataPoints *p2, int point_id, int cluster_id)
{
	double error = 0;
	for (int feature = 0; feature < p2->num_features; ++feature)
	{
		error += (p1->features_array[feature][cluster_id] - p2->features_array[feature][point_id]) * (p1->features_array[feature][cluster_id] - p2->features_array[feature][point_id]);
	}
	return error;
}

double MeanSquareError(dataPoints *point, dataPoints *centroid)
{
	double error = 0;
	for (int i = 0; i < point->num_data_points; ++i)
	{
		error += distance(centroid, point, i, point->cluster_id_of_point[i]);
	}
	return error / point->num_data_points;
}

// __device__ void distance(double *features_point, double *features_centroid, int num_features, double *distance_out)
// {
// 	*distance_out = 0;
// 	for (int feature = 0; feature < num_features; ++feature)
// 	{
// 		double tmp = features_point[feature] - features_centroid[feature];
// 		*distance_out += tmp * tmp;
// 	}
// }

__global__ void find_closest_centroids(dataPoints *points, dataPoints *centroids)
{
	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	for (int c = 0; c < centroids->num_data_points; ++c)
	{
		if (points->num_data_points < gid)
		{
			return;
		}
		int dist = 0;
		for (int feature = 0; feature < centroids->num_features; ++feature)
		{
			double tmp = points->features_array[feature][gid] - centroids->features_array[feature][c];
			dist += tmp * tmp;
		}

		if (dist < points->minDist_to_cluster[gid])
		{
			points->minDist_to_cluster[gid] = dist;
			points->cluster_id_of_point[gid] = c;
		}
	}
}

void k_means_one_iteration_gpu_thurst(dataPoints *points, dataPoints *centroids)
{
	// init

	int *nPoints = (int *)malloc(sizeof(int) * centroids->num_data_points);
	double **sum = (double **)malloc(sizeof(double *) * centroids->num_features);

	for (int feature = 0; feature < points->num_features; ++feature)
	{
		sum[feature] = (double *)malloc(sizeof(double) * centroids->num_data_points);
	}
	for (int c = 0; c < centroids->num_data_points; ++c)
	{
		nPoints[c] = 0;
		std::vector<double> tmp;

		for (int feature = 0; feature < points->num_features; ++feature)
		{
			sum[feature][c] = 0;
		}
	}

	// init

	// get nearest cluster
	int N = points->num_data_points;
	int num_threads = 1024;
	int num_blocks = (int)std::max(std::ceil((int)(N / num_threads)), 1.0);
	// size_t shmem_size = num_threads * sizeof(float);

	find_closest_centroids<<<num_blocks, num_threads>>>(points, centroids);
	hipDeviceSynchronize();

	cudaCheckError();

	thrust::device_vector<int> centroid_id_datapoint(points->num_data_points);
	thrust::copy(points->cluster_id_of_point, points->cluster_id_of_point + points->num_data_points, centroid_id_datapoint.begin());
	cudaCheckError();
	int count[5];
	for (int c = 0; c < centroids->num_data_points; c++)
	{
		count[c] = thrust::count(centroid_id_datapoint.begin(), centroid_id_datapoint.end(), c);

		cudaCheckError();
	}
	std::cout << std::endl;

	for (int feature = 0; feature < points->num_features; ++feature)
	{
		// thrust::device_vector<double> features(points->num_data_points);
		// thrust::device_vector<double> sum_position_of_centroid_featers_x(centroids->num_data_points);
		double *sumed_position = (double *)malloc(sizeof(double) * centroids->num_data_points);
		memset(sumed_position, 0, centroids->num_data_points);
		int *keys = (int *)malloc(sizeof(int) * centroids->num_data_points);
		// thrust::copy(points->features_array[feature], points->features_array[feature] + points->num_data_points-1, features.begin());
		// thrust::copy(points->features_array[feature], points->features_array[feature] + points->num_data_points, features.begin());
		thrust::sort_by_key(points->cluster_id_of_point, points->cluster_id_of_point + points->num_data_points, points->features_array[feature]);

		// auto val = features[points->num_data_points-1];
		// cudaCheckError();
		// thrust::reduce_by_key(centroid_id_datapoint.begin(), centroid_id_datapoint.end(), features.begin(), sum_position_of_centroid_featers_x.begin(), sum_position_of_centroid_featers_x.end());
		auto new_end = thrust::reduce_by_key(points->cluster_id_of_point, points->cluster_id_of_point + points->num_data_points, points->features_array[feature], keys, sumed_position);
		// std::cout<<"val:"<<val<<std::endl;
		// cudaCheckError();
		// 	for(int p =0;p<200;p++){
		// 		std::cout<<"val: "<<points->features_array[feature][p]<<", id: "<<points->cluster_id_of_point[p]<<std::endl;
		// 	}
		// 	std::cout<<std::endl;

		// std::cout<<"{ ";
		for (auto c = 0; c < centroids->num_data_points; c++)
		{
			// std::cout<<*c<<std::endl;
			centroids->features_array[feature][c] = sumed_position[c] / count[c];
		}
		// 		std::cout<<" }"<<std::endl;
	}
}

// void kMeansClusteringGPUThrust(pt *point, int epochs, int num_clusters)
// {
// 	pt *centroids = allocate_pt(point->num_features, num_clusters);
// 	cudaCheckError();
// 	// std::srand(time(0)); // need to set the random seed
// 	for (int i = 0; i < num_clusters; ++i)
// 	{
// 		int n = rand() % point->num_data_points;
// 		for (int feature = 0; feature < point->num_features; ++feature)
// 		{
// 			centroids->features_array[feature][i] = point->features_array[feature][n];
// 		}

// 		centroids->cluster_id_of_point[i] = i;
// 	}
// 	centroids->num_data_points = num_clusters;

// 	// alloc cuda memory

// 	for (int epoch = 0; epoch < epochs; ++epoch)
// 	{
// 		std::cout << "epoch: " << epoch << " Error: " << MeanSquareError(point, centroids) << std::endl;
// 		saveCsv(point, "train" + std::to_string(epoch) + ".csv");
// 		k_means_one_iteration_gpu_thurst(point, centroids);
// 	}
// }

void k_means_one_iteration_cpu(dataPoints *points, dataPoints *centroids)
{
	// init

	int *nPoints = (int *)malloc(sizeof(int) * centroids->num_data_points);
	double **sum = (double **)malloc(sizeof(double *) * centroids->num_features);

	for (int feature = 0; feature < points->num_features; ++feature)
	{
		sum[feature] = (double *)malloc(sizeof(double) * centroids->num_data_points);
	}
	for (int c = 0; c < centroids->num_data_points; ++c)
	{
		nPoints[c] = 0;
		std::vector<double> tmp;

		for (int feature = 0; feature < points->num_features; ++feature)
		{
			sum[feature][c] = 0;
		}
	}

	// init

	// get nearest cluster

	for (int p = 0; p < points->num_data_points; ++p)
	{
		for (int c = 0; c < centroids->num_data_points; ++c)
		{
			double dist = distance(centroids, points, p, c);
			if (dist < points->minDist_to_cluster[p])
			{
				points->minDist_to_cluster[p] = dist;
				points->cluster_id_of_point[p] = c;
			}
		}
	}

	// get nearest cluster

	// sum all points 'belonging' to each centroid
	for (int p = 0; p < points->num_data_points; ++p)
	{
		for (int feature = 0; feature < points->num_features; ++feature)
		{
			sum[feature][points->cluster_id_of_point[p]] += points->features_array[feature][p];
		}
		nPoints[points->cluster_id_of_point[p]]++;
	}

	// sum all points 'belonging' to each centroid

	// get centroids new location
	for (int c = 0; c < centroids->num_data_points; ++c)
	{
		for (int feature = 0; feature < points->num_features; ++feature)
		{
			centroids->features_array[feature][c] = sum[feature][c] / nPoints[c];
		}
	}
	// get centroids new location

	// find new clusters
}
void kMeansClustering(dataPoints *point, int epochs, int num_clusters, void (*k_means_one_iteration_algorithm)(dataPoints *, dataPoints *))
{
	dataPoints *centroids = allocate_pt(point->num_features, num_clusters);
	cudaCheckError();
	for (int i = 0; i < num_clusters; ++i)
	{
		int n = rand() % point->num_data_points;
		for (int feature = 0; feature < point->num_features; ++feature)
		{
			centroids->features_array[feature][i] = point->features_array[feature][n];
		}

		centroids->cluster_id_of_point[i] = i;
	}
	centroids->num_data_points = num_clusters;

	// alloc cuda memory

	for (int epoch = 0; epoch < epochs; ++epoch)
	{
		std::cout << "epoch: " << epoch << " Error: " << MeanSquareError(point, centroids) << std::endl;
		// saveCsv(point, "train" + std::to_string(epoch) + ".csv");
		k_means_one_iteration_algorithm(point, centroids);
	}
}
int main(int argc, char **argv)
{
	dataPoints *point = readCsv();
	std::srand(time(0)); // need to set the random seed

	kMeansClustering(point, 6, 5,k_means_one_iteration_gpu_thurst);
	hipFree(point);
	std::cout << "----------------\n";
	point = readCsv();
	kMeansClustering(point, 6, 5, k_means_one_iteration_cpu);

	saveCsv(point, "output.csv");

	return 0;
}
