#include "hip/hip_runtime.h"
#include "kMeansGpuThrust.h"

#include <iostream>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include "cudaCheckError.h"

#include "findClosestCentriods.h"

#include "vector"
#include "string"

void KMeansOneIterationGpuThurst(DataPoints *points, DataPoints *centroids)
{
	int N = points->num_data_points;
	int num_threads = 1024;
	int num_blocks = (int)std::max(std::ceil(((double)N / (double)num_threads)), 1.0);
	// get nearest clusters
	FindClosestCentroids<<<num_blocks, num_threads>>>(points, centroids);
	// get nearest clusters
	cudaCheckError();
	hipDeviceSynchronize();
	cudaCheckError();

	// count number of points belonging to each cluster
	int count[centroids->num_data_points];
	for (int c = 0; c < centroids->num_data_points; c++)
	{
		count[c] = thrust::count(points->cluster_id_of_point, points->cluster_id_of_point + points->num_data_points, c);
		cudaCheckError();
	}
	// count number of points belonging to each cluster

	// find new positions of the clusters
	for (int feature = 0; feature < points->num_features; ++feature)
	{
		double *sumed_position_out; 
		hipMallocManaged(&sumed_position_out, sizeof(double) * centroids->num_data_points);
		int *keys_out;
		hipMallocManaged(&keys_out, sizeof(int) * centroids->num_data_points);

		int *keys_copy;
		hipMallocManaged(&keys_copy, sizeof(int) * points->num_data_points);

		double *features_copy ;
		hipMallocManaged(&features_copy, sizeof(double) * points->num_data_points);

		for (int i = 0; i < points->num_data_points; ++i)
		{
			keys_copy[i] = points->cluster_id_of_point[i];
			features_copy[i] = points->features_array[feature][i];
		}

		thrust::sort_by_key(keys_copy, keys_copy + points->num_data_points, features_copy);

		auto new_end = thrust::reduce_by_key(keys_copy, keys_copy + points->num_data_points, features_copy, keys_out, sumed_position_out);

		for (auto c = 0; c < centroids->num_data_points; c++)
		{
			centroids->features_array[feature][c] = sumed_position_out[c] / count[c];
		}

		hipFree(keys_copy);
		hipFree(features_copy);
		hipFree(sumed_position_out);
		hipFree(keys_out);
	}
	// find new positions of the clusters
}