#include "timer.h"

// struct GpuTimer
// {
//       hipEvent_t start;
//       hipEvent_t stop;
//       float total_time;
      GpuTimer::GpuTimer()
      {
            total_time = 0;
            hipEventCreate(&start);
            hipEventCreate(&stop);
      }

      GpuTimer::~GpuTimer()
      {
            hipEventDestroy(start);
            hipEventDestroy(stop);
      }

      void GpuTimer::Start()
      {
            hipEventRecord(start, 0);
      }

      void GpuTimer::Stop()
      {
            hipEventRecord(stop, 0);
      }

      float GpuTimer::Elapsed()
      {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            total_time += elapsed;
            return elapsed;
      }
// };

GpuTimer timer_find_closest_centroids;
GpuTimer timer_compute_centroids;
GpuTimer timer_memory_allocation_gpu;
GpuTimer timer_gpu_version;
GpuTimer timer_thurst_version;
GpuTimer timer_cpu_version;

//  GpuTimer timer_closest_centroids;
//  GpuTimer timer_compute_centroids;
//  GpuTimer timer_memory_allocation;
//  GpuTimer timer_gpu_version;
//  GpuTimer timer_thurst_version;
//  GpuTimer timer_cpu_version;

// static void InitTimers()
// {
//       // timer_closest_centroids = new GpuTimer();
//       // timer_compute_centroids = new GpuTimer();
//       // timer_memory_allocation = new GpuTimer();
//       // timer_gpu_version = new GpuTimer();
//       // timer_thurst_version = new GpuTimer();
//       // timer_cpu_version = new GpuTimer();
// }
