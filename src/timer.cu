#include "timer.h"
#include "cudaCheckError.h"
// struct GpuTimer
// {
//       hipEvent_t start;
//       hipEvent_t stop;
//       float total_time;
GpuTimer::GpuTimer()
{
      total_time = 0;
      hipEventCreate(&start);
      cudaCheckError();

      hipEventCreate(&stop);
      cudaCheckError();
}

GpuTimer::~GpuTimer()
{
      hipEventDestroy(start);
      cudaCheckError();

      hipEventDestroy(stop);
      cudaCheckError();
}

void GpuTimer::Start()
{
      hipEventRecord(start, 0);
      cudaCheckError();
}

void GpuTimer::Stop()
{
      hipEventRecord(stop, 0);
      cudaCheckError();
}

float GpuTimer::Elapsed()
{
      float elapsed;
      hipEventSynchronize(stop);
      cudaCheckError();
      hipEventElapsedTime(&elapsed, start, stop);
      cudaCheckError();

      total_time += elapsed;
      return elapsed;
}
// };

GpuTimer *timer_find_closest_centroids;
GpuTimer *timer_compute_centroids;
GpuTimer *timer_memory_allocation_gpu;
GpuTimer *timer_gpu_version;
GpuTimer *timer_thurst_version;
GpuTimer *timer_cpu_version;

//  GpuTimer timer_closest_centroids;
//  GpuTimer timer_compute_centroids;
//  GpuTimer timer_memory_allocation;
//  GpuTimer timer_gpu_version;
//  GpuTimer timer_thurst_version;
//  GpuTimer timer_cpu_version;
