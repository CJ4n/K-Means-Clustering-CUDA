#include "hip/hip_runtime.h"
#include "KMeansGpu.h"

#include "hip/hip_runtime.h"

#include "Config.h"
#include "CudaCheckError.h"
#include "FindClosestCentriods.h"
#include "Timer.h"

// kernels

#define INDEX(f, c, tid, num_clusters, num_features) ((f * num_clusters) + c) + tid *(num_features + 1) * num_clusters
#define INDEX_ID(c, tid, num_clusters, num_features) ((num_features * num_clusters) + c) + tid *(num_features + 1) * num_clusters
template <int F_NUM>
__global__ void ReduceDataPoints(MyDataType **features, int *cluster_ids, MyDataType **centroids_features,
								 const int count_in, CountType *count_out, const int num_data_points, const int num_clusters, int act)
{
	extern __shared__ MyDataType shm[];
	const int tid = threadIdx.x;
	const int gid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	if (gid >= num_data_points)
	{
		return;
	}
	// shm[(f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5},{count1,...,count5}), (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5},{count1,...,count5}),..., (f1{c1,c2,c3,c4,c5},f2{c1,c2,c3,c4,c5},{count1,...,count5}) ]

	int c1, c2;

	for (int f = 0; f < F_NUM; ++f)
	{
		for (int c = 0; c < num_clusters; ++c)
		{
			shm[INDEX(f, c, tid, num_clusters, F_NUM)] = 0;
			shm[INDEX_ID(c, tid, num_clusters, F_NUM)] = 0;
		}

		c1 = cluster_ids[gid];
		shm[INDEX(f, c1, tid, num_clusters, F_NUM)] += features[f][gid];

		if (gid + blockDim.x >= num_data_points)
		{
			continue;
		}

		c2 = cluster_ids[gid + blockDim.x];
		shm[INDEX(f, c2, tid, num_clusters, F_NUM)] += features[f][gid + blockDim.x];
		// idx where to store particualr feature coord
	}
	{
		if (count_in)
			shm[INDEX_ID(c1, tid, num_clusters, F_NUM)] = count_in;
		else
			shm[INDEX_ID(c1, tid, num_clusters, F_NUM)] = count_out[gid];

		if (gid + blockDim.x < num_data_points)
		{
			if (count_in)
				shm[INDEX_ID(c2, tid, num_clusters, F_NUM)] += count_in;
			else
				shm[INDEX_ID(c2, tid, num_clusters, F_NUM)] += count_out[gid + blockDim.x];
		}
	}
	__syncthreads();

	for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2)
	{
		if (tid < stride)
		{
			for (int f = 0; f < F_NUM; ++f)
			{
				for (int c = 0; c < num_clusters; ++c)
				{
					shm[INDEX(f, c, tid, num_clusters, F_NUM)] += shm[INDEX(f, c, (tid + stride), num_clusters, F_NUM)];
					if (f == 0)
					{
						shm[INDEX_ID(c, tid, num_clusters, F_NUM)] += shm[INDEX_ID(c, (tid + stride), num_clusters, F_NUM)];
					}
				}
			}
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		for (int f = 0; f < F_NUM; ++f)
			for (int c = 0; c < num_clusters; ++c)
			{
				{
					centroids_features[f][c + blockIdx.x * num_clusters] = shm[INDEX(f, c, 0, num_clusters, F_NUM)];
					if (f == 0)
					{
						// [{count1,...,count5},{count1,...,count5},..,
						count_out[blockIdx.x * num_clusters + c] = shm[INDEX_ID(c, 0, num_clusters, F_NUM)];
					}
				}
			}
	}
}

#define INDEX_C(c, tid, num_clusters) c + (tid * num_clusters)
__device__ void warpReduceCount(volatile CountType *shm, int tid, int c, int num_clusters)
{
	shm[INDEX_C(c, tid, num_clusters)] += shm[INDEX_C(c, (tid + 32), num_clusters)];
	shm[INDEX_C(c, tid, num_clusters)] += shm[INDEX_C(c, (tid + 16), num_clusters)];
	shm[INDEX_C(c, tid, num_clusters)] += shm[INDEX_C(c, (tid + 8), num_clusters)];
	shm[INDEX_C(c, tid, num_clusters)] += shm[INDEX_C(c, (tid + 4), num_clusters)];
	shm[INDEX_C(c, tid, num_clusters)] += shm[INDEX_C(c, (tid + 2), num_clusters)];
	shm[INDEX_C(c, tid, num_clusters)] += shm[INDEX_C(c, (tid + 1), num_clusters)];
}
__global__ void ReduceDataPointsCountPoints(const int *cluster_ids,
											const CountType count_in, CountType *count_out, const int num_data_points, const int num_clusters, int active_threads_count)
{
	extern __shared__ CountType shm_c[];
	const int tid = threadIdx.x;
	const int gid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	int c1, c2;

	for (int c = 0; c < num_clusters; ++c)
	{
		shm_c[INDEX_C(c, tid, num_clusters)] = 0;
	}
	if (gid >= num_data_points)
	{
		return;
	}
	if (tid >= active_threads_count)
	{
		return;
	}
	c1 = cluster_ids[gid];
	if (gid + active_threads_count < num_data_points)
	{
		c2 = cluster_ids[gid + active_threads_count];
	}

	if (count_in)
		shm_c[INDEX_C(c1, tid, num_clusters)] = count_in;
	else
		shm_c[INDEX_C(c1, tid, num_clusters)] = count_out[gid];

	if (gid + active_threads_count < num_data_points)
	{
		if (count_in)
			shm_c[INDEX_C(c2, tid, num_clusters)] += count_in;
		else
			shm_c[INDEX_C(c2, tid, num_clusters)] += count_out[gid + active_threads_count];
	}
	__syncthreads();

	for (unsigned int stride = blockDim.x / 2; stride > 32; stride /= 2)
	{
		if (tid < stride)
		{
			for (int c = 0; c < num_clusters; ++c)
			{
				shm_c[INDEX_C(c, tid, num_clusters)] += shm_c[INDEX_C(c, (tid + stride), num_clusters)];
			}
		}
		__syncthreads();
	}
	if (tid < 32)
	{
		for (int c = 0; c < num_clusters; ++c)
		{
			warpReduceCount(shm_c, tid, c, num_clusters);
		}
	}
	__syncthreads();

	if (tid == 0)
	{
		for (int c = 0; c < num_clusters; ++c)
		{
			count_out[blockIdx.x * num_clusters + c] = shm_c[INDEX_C(c, 0, num_clusters)];
		}
	}
}

#define INDEX_F(c, tid, num_clusters) c + (tid * num_clusters)
__device__ void warpReduceFeature(volatile MyDataType *shm, int tid, int c, int num_clusters)
{
	shm[INDEX_F(c, tid, num_clusters)] += shm[INDEX_F(c, (tid + 32), num_clusters)];
	shm[INDEX_F(c, tid, num_clusters)] += shm[INDEX_F(c, (tid + 16), num_clusters)];
	shm[INDEX_F(c, tid, num_clusters)] += shm[INDEX_F(c, (tid + 8), num_clusters)];
	shm[INDEX_F(c, tid, num_clusters)] += shm[INDEX_F(c, (tid + 4), num_clusters)];
	shm[INDEX_F(c, tid, num_clusters)] += shm[INDEX_F(c, (tid + 2), num_clusters)];
	shm[INDEX_F(c, tid, num_clusters)] += shm[INDEX_F(c, (tid + 1), num_clusters)];
}
__global__ void ReduceDataPointsByFeatures(MyDataType *features, const int *cluster_ids, MyDataType *out,
										   const int num_data_points, const int num_clusters, int active_threads_count)
{
	extern __shared__ MyDataType shm_f[];
	const int tid = threadIdx.x;
	const int gid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	int c1, c2;

	for (int c = 0; c < num_clusters; ++c)
	{
		shm_f[INDEX_F(c, tid, num_clusters)] = 0;
	}
	if (gid >= num_data_points)
	{
		return;
	}
	if (tid >= active_threads_count)
	{
		return;
	}
	c1 = cluster_ids[gid];
	if (gid + active_threads_count < num_data_points)
	{
		c2 = cluster_ids[gid + active_threads_count];
	}

	shm_f[INDEX_F(c1, tid, num_clusters)] = features[gid];
	if (gid + active_threads_count < num_data_points)
	{
		shm_f[INDEX_F(c2, tid, num_clusters)] += features[gid + active_threads_count];
	}
	__syncthreads();

	for (unsigned int stride = blockDim.x / 2; stride > 32; stride /= 2)
	{
		if (tid < stride)
		{
			for (int c = 0; c < num_clusters; ++c)
			{
				shm_f[INDEX_F(c, tid, num_clusters)] += shm_f[INDEX_F(c, (tid + stride), num_clusters)];
			}
		}
		__syncthreads();
	}

	if (tid < 32)
	{
		for (int c = 0; c < num_clusters; ++c)
		{
			warpReduceFeature(shm_f, tid, c, num_clusters);
		}
	}
	__syncthreads();
	if (tid == 0)
	{
		for (int c = 0; c < num_clusters; ++c)
		{
			out[c + blockIdx.x * num_clusters] = shm_f[INDEX_F(c, 0, num_clusters)];
		}
	}
}

__global__ void FindNewCentroids(DataPoints *centroids, CountType *count, DataPoints *reduced_points)
{
	const int f = threadIdx.x;
	const int c = threadIdx.y;
	centroids->features_array[f][c] = reduced_points->features_array[f][c] / (MyDataType)count[c];
}

__global__ void InitPointsWithCentroidsIds(DataPoints *points, int num_clusters, int num_points)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	if (gid >= num_points)
	{
		return;
	}
	points->cluster_id_of_point[gid] = gid % num_clusters;
}

// kernels

#define MAX_SHM_SIZE 48 * 1024
#define DEFAULT_NUM_THREADS 1024l
#define CALCULATE_SHM_SIZE_JOIN_REDUCE(num_features, num_clusters, num_threads) num_threads *(num_features + 1) * num_clusters * sizeof(MyDataType)
#define CALCULATE_SHM_SIZE_FEATURE_WISE_REDUCE(num_clusters, num_threads) num_threads *num_clusters * sizeof(MyDataType)
#define CALCULATE_SHM_SIZE_COUNT(num_clusters, num_threads) num_threads *num_clusters * sizeof(CountType)

DataPoints *reduced_points;
CountType *ids_count;
int cur_epoch = 0;

// #define NUM_STREAM (NUM_FEATURES + 1)
// hipStream_t streams[NUM_STREAM];
template <int F_NUM>
void debugFunction(DataPoints *points, CountType *ids_count, int num_clusters, int num_blocks, int num_threads, int N, std::string label)
{
	std::cout << "\n---------" << label << "---------" << std::endl;
	DataPoints *debug = AllocateDataPoints(F_NUM, num_clusters);

	MyDataType sum_tot = 0;
	// Gets exact sum by feature and clusters
	for (int f = 0; f < F_NUM; f++)
		for (int c = 0; c < num_clusters; c++)
		{
			debug->features_array[f][c] = 0;
		}

	for (int i = 0; i < points->num_data_points; i++)
	{
		for (int f = 0; f < F_NUM; f++)
		{
			debug->features_array[f][points->cluster_id_of_point[i]] += points->features_array[f][i];
			sum_tot += points->features_array[f][i];
		}
	}

	std::cout << " correct (points)\n{\n	";
	double sum_tot_v2 = 0;
	for (int c = 0; c < num_clusters; c++)
	{
		for (int f = 0; f < F_NUM; f++)
		{
			std::cout << debug->features_array[f][c] << ", ";
			sum_tot_v2 += debug->features_array[f][c];
			debug->features_array[f][c] = 0;
		}
	}
	std::cout << "\n}\n";
	// Gets exact sum by feature and clusters

	// Gets redcued sum by feature and cluster
	MyDataType sum_tot_reduced = 0;

	for (int i = 0; i < num_blocks * num_clusters; i++)
	{
		for (int f = 0; f < F_NUM; f++)
		{
			debug->features_array[f][reduced_points->cluster_id_of_point[i]] += reduced_points->features_array[f][i];
			sum_tot_reduced += reduced_points->features_array[f][i];
		}
	}
	std::cout << "Calculated points (reduced_points)\n{\n	";

	double sum_tot_reduced_v2 = 0;
	for (int c = 0; c < num_clusters; c++)
	{
		for (int f = 0; f < F_NUM; f++)
		{
			std::cout << debug->features_array[f][c] << ", ";
			sum_tot_reduced_v2 += debug->features_array[f][c];
			debug->features_array[f][c] = 0;
		}
	}
	std::cout << "\n}\n";

	// Gets redcued sum by feature and cluster

	std::cout << "sumed all points(sum_tot):           " << sum_tot << std::endl;
	std::cout << "sumed all points(sum_tot_v2)         " << sum_tot_v2 << std::endl;
	std::cout << "sumed all points(sum_tot_reduced)    " << sum_tot_reduced << std::endl;
	std::cout << "sumed all points(sum_tot_reduced_v2) " << sum_tot_reduced_v2 << std::endl;

	CountType *count_check = (CountType *)malloc(sizeof(CountType) * num_clusters);
	memset(count_check, 0, sizeof(CountType) * num_clusters);

	// Gets exact count of ids
	CountType exact_points_count = 0;
	for (int i = 0; i < points->num_data_points; ++i)
	{
		count_check[points->cluster_id_of_point[i]]++;
		// std::cout<<points->cluster_id_of_point[i]<<", ";
	}
	std::cout << "Exact ids count\n{\n	";
	for (int c = 0; c < num_clusters; ++c)
	{
		std::cout << count_check[c] << ", ";
		exact_points_count += count_check[c];
	}
	std::cout << "\n}\n";
	// Gets exact count of ids

	memset(count_check, 0, sizeof(CountType) * num_clusters);

	// Gets reduced count of ids
	CountType reduced_points_count = 0;
	for (int i = 0; i < num_blocks; ++i)
	{
		for (int c = 0; c < num_clusters; ++c)
		{
			count_check[c] += ids_count[i * num_clusters + c];
			reduced_points_count += ids_count[i * num_clusters + c];
		}
	}
	std::cout << "Reduced ids count\n{\n	";
	for (int c = 0; c < num_clusters; ++c)
	{
		std::cout << count_check[c] << ", ";
	}
	std::cout << "\n}\n";
	// Gets reduced count of ids

	std::cout << "number of points (exact_points_count):   " << exact_points_count << std::endl;
	std::cout << "number of points (reduced_points_count): " << reduced_points_count << std::endl;

	free(count_check);
	if (num_blocks != -1)
		if (num_blocks * num_threads * 2 < N || N != num_clusters * num_blocks)
		{
			std::cout << "aaaaaaaaaaaaaaaaaaaaaa\n";
		}
	DeallocateDataPoints(debug);
	cudaCheckError();
}

int GetNumBlocks(int num_threads, int cur_num_blocks, int num_clusters)
{
	int N = cur_num_blocks * num_clusters;
	int num_blocks = std::ceil((float)N / (float)num_threads / 2.0);
	return num_blocks;
}

template <int F_NUM>
void ReduceFeature(DataPoints *points, DataPoints *out, CountType *ids_count, int num_clusters,
				   int N, CountType count_in, int *num_th, int *num_bl, int atc)
{
	int num_threads = *num_th;
	int num_blocks = *num_bl;
	if (RUN_REDUCE_FEATURE_WISE)
	{
		size_t shm_size = CALCULATE_SHM_SIZE_FEATURE_WISE_REDUCE(num_clusters, num_threads);
		for (int f = 0; f < F_NUM; ++f)
		{
			// ReduceDataPointsByFeatures<<<num_blocks, num_threads, shm_size,streams[f]>>>(points->features_array[f],
			// 																  points->cluster_id_of_point, out->features_array[f],
			// 																  N, num_clusters, atc);
			ReduceDataPointsByFeatures<<<num_blocks, num_threads, shm_size>>>(points->features_array[f],
																			  points->cluster_id_of_point, out->features_array[f],
																			  N, num_clusters, atc);
			cudaCheckError();
		}

		shm_size = CALCULATE_SHM_SIZE_COUNT(num_clusters, num_threads);
		// ReduceDataPointsCountPoints<<<num_blocks, num_threads, shm_size,streams[NUM_STREAM-1]>>>(points->cluster_id_of_point,
		// 																   count_in, ids_count, N, num_clusters, atc);
		ReduceDataPointsCountPoints<<<num_blocks, num_threads, shm_size>>>(points->cluster_id_of_point,
																		   count_in, ids_count, N, num_clusters, atc);
	}
	else if (!RUN_REDUCE_FEATURE_WISE)
	{
		size_t shm_size = CALCULATE_SHM_SIZE_JOIN_REDUCE(F_NUM, num_clusters, num_threads);
		ReduceDataPoints<F_NUM><<<num_blocks, num_threads, shm_size>>>(points->features_array, points->cluster_id_of_point, reduced_points->features_array, count_in, ids_count, N, num_clusters, atc);
	}
}

template <int N_FEATURES>
MyDataType KMeansOneIterationGpu(DataPoints *points, DataPoints *centroids)
{

	// for (int i = 0; i < NUM_STREAM; i++)
	// {
	// 	hipStreamCreate(&streams[i]);
	// }

	const int num_clusters = centroids->num_data_points;
	int N = points->num_data_points;

	// Find closest centroids for each datapoint
	const int num_threads_find_closest = 1024;
	const int num_blocks_find_closest = std::max(1, (int)std::ceil(points->num_data_points / num_threads_find_closest));
	const size_t shm_find_closest = sizeof(MyDataType) * num_clusters * N_FEATURES + sizeof(MyDataType) * num_threads_find_closest * N_FEATURES;
	if (MEASURE_TIME)
	{
		timer_find_closest_centroids->Start();
	}
	FindClosestCentroids<N_FEATURES><<<num_blocks_find_closest, num_threads_find_closest, shm_find_closest>>>(points->features_array,
																											  points->cluster_id_of_point, centroids->features_array, points->num_data_points,
																											  N_FEATURES, num_clusters);
	if (SYNCHRONIZE_AFTER_KERNEL_RUN)
	{
		hipDeviceSynchronize();
	}
	if (MEASURE_TIME)
	{
		timer_find_closest_centroids->Stop();
		timer_find_closest_centroids->Elapsed();
	}
	cudaCheckError();
	// Find closest centroids for each datapoint

	// Create and init reduced points, what will be used sum up all points
	int num_threads = DEFAULT_NUM_THREADS;

	if (!RUN_REDUCE_FEATURE_WISE)
	{
		while (MAX_SHM_SIZE < CALCULATE_SHM_SIZE_JOIN_REDUCE(N_FEATURES, num_clusters, num_threads))
		{
			num_threads /= 2;
		}
	}
	else
	{
		while (MAX_SHM_SIZE < CALCULATE_SHM_SIZE_FEATURE_WISE_REDUCE(num_clusters, num_threads))
		{
			num_threads /= 2;
		}
		while (MAX_SHM_SIZE < CALCULATE_SHM_SIZE_COUNT(num_clusters, num_threads))
		{
			num_threads /= 2;
		}
	}

	int num_blocks = (int)std::max(std::ceil((long)(N / (double)num_threads / 2)), 1.0);

	const long num_reduced_points = num_blocks * num_clusters;

	if (cur_epoch == 0)
	{
		int num_threads_inti_id = (int)std::min(DEFAULT_NUM_THREADS, num_reduced_points);
		int num_block_init_id = (int)std::max(std::ceil((num_reduced_points / (double)num_threads_inti_id)), 1.0);
		reduced_points = AllocateDataPoints(N_FEATURES, num_reduced_points);
		InitPointsWithCentroidsIds<<<num_block_init_id, num_threads_inti_id>>>(reduced_points, num_clusters, num_reduced_points);
		cudaCheckError();
		hipMallocManaged(&ids_count, sizeof(CountType) * num_reduced_points);
		cudaCheckError();
		hipMemset(ids_count, 0, sizeof(CountType) * num_reduced_points);
		cudaCheckError();
	}
	else
	{
		hipMemset(ids_count, 0, sizeof(CountType) * num_reduced_points);
		cudaCheckError();
	}
	// Create and init reduced points

	// reduce points in `points` and store them in reduced_poitsn
	if (MEASURE_TIME)
	{
		timer_compute_centroids->Start();
	}
	ReduceFeature<N_FEATURES>(points, reduced_points, ids_count, num_clusters, N, 1, &num_threads, &num_blocks, num_threads);
	if (SYNCHRONIZE_AFTER_KERNEL_RUN)
	{
		hipDeviceSynchronize();
	}
	if (MEASURE_TIME)
	{
		timer_compute_centroids->Stop();
		timer_compute_centroids->Elapsed();
	}
	cudaCheckError();
	// reduce points in `points` and store them in reduced_poitsn
	if (DEBUG_GPU_ITERATION)
	{
		debugFunction<N_FEATURES>(points, ids_count, num_clusters, num_blocks, num_threads, num_clusters * num_blocks, "BEFORE WHILE REDUCE");
	}

	// further reduce points in `reduced_points`, until there will be no more then  `num_threads * 2` poitns left to reduce
	while (num_blocks * num_clusters > num_threads * 2)
	{
		N = num_blocks * num_clusters;
		num_blocks = GetNumBlocks(num_threads, num_blocks, num_clusters);

		if (MEASURE_TIME)
		{
			timer_compute_centroids->Start();
		}
		ReduceFeature<N_FEATURES>(reduced_points, reduced_points, ids_count, num_clusters, N, 0, &num_threads, &num_blocks, num_threads);
		if (SYNCHRONIZE_AFTER_KERNEL_RUN)
		{
			hipDeviceSynchronize();
		}
		if (MEASURE_TIME)
		{
			timer_compute_centroids->Stop();
			timer_compute_centroids->Elapsed();
		}
		cudaCheckError();
	}
	if (DEBUG_GPU_ITERATION)
	{
		debugFunction<N_FEATURES>(points, ids_count, num_clusters, num_blocks, num_threads, num_blocks * num_clusters, "AFTER WHILE REDUCE");
	}
	// further reduce points in `reduced_points`, until there will be no more then  `num_threads * 2` poitns left to reduce

	// last reduce, reduce all remaining points to a 'single datapoint', that is: points belonging to the same cluster will be reduced to single point
	if (num_blocks > 1) // if may happen, that last reduced reduced all the point, happens when num_blocks==1
	{
		N = num_clusters * num_blocks;
		num_blocks = GetNumBlocks(num_threads, num_blocks, num_clusters);
		if (MEASURE_TIME)
		{
			timer_compute_centroids->Start();
		}
		ReduceFeature<N_FEATURES>(reduced_points, reduced_points, ids_count, num_clusters, N, 0, &num_threads, &num_blocks, std::ceil(N / 2.0));
		if (SYNCHRONIZE_AFTER_KERNEL_RUN)
		{
			hipDeviceSynchronize();
		}

		if (MEASURE_TIME)
		{
			timer_compute_centroids->Stop();
			timer_compute_centroids->Elapsed();
		}
		cudaCheckError();
		if (DEBUG_GPU_ITERATION)
		{
			debugFunction<N_FEATURES>(points, ids_count, num_clusters, 1, num_threads, 1 * num_clusters, "AFTER LAST REDUCE");
		}
	}
	// last reduce, reduce all remaining points

	// find new centroids
	dim3 grid(1, 1, 1);
	dim3 block(N_FEATURES, num_clusters);
	FindNewCentroids<<<grid, block>>>(centroids, ids_count, reduced_points);
	cudaCheckError();
	// find new centroids

	// clean up, if it is last epoch
	if (cur_epoch + 1 == NUM_EPOCHES)
	{
		DeallocateDataPoints(reduced_points);
		cudaCheckError();
		hipFree(ids_count);
		cudaCheckError();
		cur_epoch = 0;
	}
	else
	{
		cur_epoch++;
	}
	// clean up, if it is last epoch

	// for (int i = 0; i < NUM_STREAM; i++)
	// {
	// 	hipStreamDestroy(streams[i]);
	// }
	return MeanSquareErrorParallel<N_FEATURES>(points, centroids);
}

template MyDataType KMeansOneIterationGpu<NUM_FEATURES>(DataPoints *points, DataPoints *centroids);