#include "hip/hip_runtime.h"
#include "FindClosestCentriods.h"

#include "Config.h"

#define INDEX_CLUSTER(f, c, num_clusters) ((f * num_clusters) + c)
#define INDEX_POINT(f, tid, num_clusters, num_features) (num_features * num_clusters + tid * num_features + f)
template <int F_NUM>
__global__ void FindClosestCentroids(MyDataType **features, int *centroids_ids, MyDataType **centeriods_features, const int num_points, const int num_features, const int num_clusters)
{
	//  centroids				| data points
	// (f1{c1,c2,c3}f2{c1,c2,c3}|f1{c1,c2,c3}f2{c1,c2,c3},...,f1{c1,c2,c3}f2{c1,c2,c3})
	extern __shared__ MyDataType shm2[];
	const int tid = threadIdx.x;
	const int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid >= num_points)
	{
		return;
	}

	if (tid < num_clusters * F_NUM)
	{
		int x = tid / num_clusters;
		int y = tid % num_clusters;
		shm2[INDEX_CLUSTER(x, y, num_clusters)] = centeriods_features[x][y];
	}

	for (int f = 0; f < F_NUM; ++f)
	{
		shm2[INDEX_POINT(f, tid, num_clusters, F_NUM)] = features[f][gid];
	}
	MyDataType min_dist = __DBL_MAX__;

	__syncthreads();
	int cur_centroids = -1;

	for (int c = 0; c < num_clusters; ++c)
	{
		MyDataType dist = 0;
		for (int f = 0; f < F_NUM; ++f)
		{
			MyDataType tmp = shm2[INDEX_POINT(f, tid, num_clusters, F_NUM)] - shm2[INDEX_CLUSTER(f, c, num_clusters)];
			dist += tmp * tmp;
		}

		if (dist < min_dist)
		{
			min_dist = dist;
			cur_centroids = c;
		}
	}

	centroids_ids[gid] = cur_centroids;
}

template __global__ void FindClosestCentroids<NUM_FEATURES>(MyDataType **features, int *centroids_ids, MyDataType **centeriods_features, const int num_points, const int num_features, const int num_clusters);
