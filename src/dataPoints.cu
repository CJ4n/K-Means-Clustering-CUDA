#pragma once

#include <hip/hip_runtime.h>
#include <fstream>	// for file-reading
#include <iostream> // for file-reading
#include <sstream>	// for file-reading
#include <vector>
#include <dataPoints.h>
#include "cudaCheckError.h"

DataPoints *AllocateDataPoints(int num_features, int num_data_points)
{

	DataPoints *point;
	hipMallocManaged(&point, sizeof(DataPoints));
	cudaCheckError();

	point->num_data_points = num_data_points;
	hipMallocManaged(&(point->cluster_id_of_point), sizeof(int) * num_data_points);
	cudaCheckError();
	// for(int i = 0;i<num_data_points;i++){
	// 	point->cluster_id_of_point[i]=1;
	// }
	hipMallocManaged(&(point->minDist_to_cluster), sizeof(float) * num_data_points);
	cudaCheckError();

	point->num_features = num_features;
	hipMallocManaged(&(point->features_array), sizeof(float *) * point->num_features);
	cudaCheckError();

	for (int feature = 0; feature < point->num_features; ++feature)
	{
		hipMallocManaged(&(point->features_array[feature]), sizeof(*(point->features_array[feature])) * point->num_data_points);
		cudaCheckError();
	}
	return point;
}

void DeallocateDataPoints(DataPoints *data_points )
{
	for (int f = 0; f < data_points->num_features; f++)
	{
		hipFree(data_points->features_array[f]);
	}
	hipFree(data_points->features_array);
	hipFree(data_points->cluster_id_of_point);
	hipFree(data_points->minDist_to_cluster);
	hipFree(data_points);
}

float Distance(DataPoints *p1, DataPoints *p2, int point_id, int cluster_id)
{
	float error = 0;
	for (int feature = 0; feature < p2->num_features; ++feature)
	{
		error += (p1->features_array[feature][cluster_id] - p2->features_array[feature][point_id]) * (p1->features_array[feature][cluster_id] - p2->features_array[feature][point_id]);
	}
	return error;
}

float MeanSquareError(DataPoints *point, DataPoints *centroid)
{
	float error = 0;
	for (int i = 0; i < point->num_data_points; ++i)
	{
		error += Distance(centroid, point, i, point->cluster_id_of_point[i]);
	}
	return error / point->num_data_points;
}

DataPoints *ReadCsv()
{
	std::vector<Point> points;
	std::string line;
	std::ifstream file("/home/jan/Desktop/K-Means-Clustering-CUDA/mall_data.csv");
	// std::ifstream file("../mall_data.csv");
	while (std::getline(file, line))
	{
		std::stringstream lineStream(line);
		std::string bit;
		float x, y;
		getline(lineStream, bit, ',');
		x = std::stof(bit);
		getline(lineStream, bit, '\n');
		y = stof(bit);

		points.push_back(Point(x, y));
	}
	file.close();

	DataPoints *point = AllocateDataPoints(2, points.size());
	int i = 0;
	for (std::vector<Point>::iterator it = points.begin(); it != points.end(); ++it)
	{
		float XY[2];
		XY[0] = it->x;
		XY[1] = it->y;
		for (int feature = 0; feature < point->num_features; ++feature)
		{
			point->features_array[feature][i] = XY[feature];
		}
		point->cluster_id_of_point[i] = it->cluster;
		point->minDist_to_cluster[i] = __FLT_MAX__;
		i++;
	}
	return point;
}

void SaveCsv(DataPoints *point, std::string file_name)
{
	std::ofstream myfile;
	std::remove(file_name.c_str());
	myfile.open(file_name);
	myfile << "x,y,c" << std::endl;

	for (int i = 0; i < point->num_data_points; ++i)
	{
		for (int feature = 0; feature < point->num_features; ++feature)
		{
			myfile << point->features_array[feature][i];
			myfile << ",";
		}

		myfile << point->cluster_id_of_point[i] << std::endl;
	}
	
	myfile.close();
}