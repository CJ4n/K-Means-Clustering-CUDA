#include <hip/hip_runtime.h>
#include <fstream>	// for file-reading
#include <iostream> // for file-reading
#include <sstream>	// for file-reading
#include <vector>
#include <dataPoints.h>
#include "cudaCheckError.h"
#include "timer.h"
#include "Constants.h"
DataPoints *AllocateDataPoints(int num_features, int num_data_points, const bool malloc_managed)
{
	if (MEASURE_TIME)
	{
		timer_data_generations->Start();
	}
	// if (malloc_managed == false)
	// {
	// 	DataPoints *point;
	// 	hipMalloc(&point, sizeof(DataPoints));
	// 	cudaCheckError();
	// 	int *tmp1 =(int* ) malloc(sizeof(int));
	// 	*tmp1 = num_data_points;

	// 	hipMemcpy(&(point->num_data_points), tmp1, sizeof(int), hipMemcpyHostToDevice);
	// 	// point->num_data_points = num_data_points;
	// 	cudaCheckError();

	// 	hipMalloc(&(point->cluster_id_of_point), sizeof(int) * num_data_points);
	// 	cudaCheckError();
	// 	hipMemset(point->cluster_id_of_point, 0, sizeof(int) * num_data_points);
	// 	cudaCheckError();

	// 	// point->num_features = num_features;
	// 	int *tmp2 =(int* ) malloc(sizeof(int));
	// 	*tmp2= num_features;
	// 	hipMemcpy(&(point->num_features), tmp2, sizeof(int), hipMemcpyHostToDevice);
	// 	cudaCheckError();

	// 	hipMalloc(&(point->features_array), sizeof(*(point->features_array)) * point->num_features);
	// 	cudaCheckError();

	// 	for (int feature = 0; feature < point->num_features; ++feature)
	// 	{
	// 		hipMalloc(&(point->features_array[feature]), sizeof(MyDataType) * point->num_data_points);
	// 		cudaCheckError();
	// 		hipMemset(point->features_array[feature], 0, sizeof(MyDataType) * point->num_data_points);
	// 		cudaCheckError();
	// 	}
	// 	free(tmp1);
	// 	free(tmp2);
	// 	return point;
	// }
	// else
	// {

	DataPoints *point;
	hipMallocManaged(&point, sizeof(DataPoints));
	cudaCheckError();

	point->num_data_points = num_data_points;
	hipMallocManaged(&(point->cluster_id_of_point), sizeof(int) * num_data_points);
	cudaCheckError();
	hipMemset(point->cluster_id_of_point, 0, sizeof(int) * num_data_points);
	cudaCheckError();

	point->num_features = num_features;
	hipMallocManaged(&(point->features_array), sizeof(*(point->features_array)) * point->num_features);
	cudaCheckError();

	for (int feature = 0; feature < point->num_features; ++feature)
	{
		hipMallocManaged(&(point->features_array[feature]), sizeof(MyDataType) * point->num_data_points);
		cudaCheckError();
		hipMemset(point->features_array[feature], 0, sizeof(MyDataType) * point->num_data_points);
		cudaCheckError();
	}
	if (MEASURE_TIME)
	{
		timer_data_generations->Stop();
		timer_data_generations->Elapsed();
	}
	return point;
	// }
}

void DeallocateDataPoints(DataPoints *data_points)
{
	if (MEASURE_TIME)
	{
		timer_data_generations->Start();
	}
	for (int f = 0; f < data_points->num_features; f++)
	{
		hipFree(data_points->features_array[f]);
	}
	hipFree(data_points->features_array);
	hipFree(data_points->cluster_id_of_point);
	hipFree(data_points);
	if (MEASURE_TIME)
	{
		timer_data_generations->Stop();
		timer_data_generations->Elapsed();
	}
}

MyDataType Distance(const DataPoints *p1, const DataPoints *p2, const int point_id, const int cluster_id)
{
	MyDataType error = 0;
	for (int feature = 0; feature < p2->num_features; ++feature)
	{
		error += (p1->features_array[feature][cluster_id] - p2->features_array[feature][point_id]) * (p1->features_array[feature][cluster_id] - p2->features_array[feature][point_id]);
	}
	return error;
}
#include <unistd.h>
MyDataType MeanSquareError(const DataPoints *point, const DataPoints *centroid)
{
	MyDataType error = 0;
	for (int i = 0; i < point->num_data_points; ++i)
	{
		error += Distance(centroid, point, i, point->cluster_id_of_point[i]);
	}
	return error / point->num_data_points;
}

// DataPoints *ReadCsv()
// {
// 	std::vector<Point> points;
// 	std::string line;
// 	std::ifstream file("/home/jan/Desktop/K-Means-Clustering-CUDA/mall_data.csv");
// 	// std::ifstream file("../mall_data.csv");
// 	while (std::getline(file, line))
// 	{
// 		std::stringstream lineStream(line);
// 		std::string bit;
// 		float x, y;
// 		getline(lineStream, bit, ',');
// 		x = std::stof(bit);
// 		getline(lineStream, bit, '\n');
// 		y = stof(bit);

// 		points.push_back(Point(x, y));
// 	}
// 	file.close();

// 	DataPoints *point = AllocateDataPoints(2, points.size());
// 	int i = 0;
// 	for (std::vector<Point>::iterator it = points.begin(); it != points.end(); ++it)
// 	{
// 		float XY[2];
// 		XY[0] = it->x;
// 		XY[1] = it->y;
// 		for (int feature = 0; feature < point->num_features; ++feature)
// 		{
// 			point->features_array[feature][i] = XY[feature];
// 		}
// 		point->cluster_id_of_point[i] = it->cluster;
// 		i++;
// 	}
// 	return point;
// }

void SaveCsv(const DataPoints *point, const std::string file_name)
{
	std::ofstream myfile;
	std::remove(file_name.c_str());
	myfile.open(file_name);
	myfile << "x,y,c" << std::endl;

	for (int i = 0; i < point->num_data_points; ++i)
	{
		for (int feature = 0; feature < point->num_features; ++feature)
		{
			myfile << point->features_array[feature][i];
			myfile << ",";
		}

		myfile << point->cluster_id_of_point[i] << std::endl;
	}

	myfile.close();
}