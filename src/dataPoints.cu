#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fstream>	// for file-reading
#include <iostream> // for file-reading
#include <sstream>	// for file-reading
#include <vector>
#include <dataPoints.h>
#include "cudaCheckError.h"
#include "timer.h"
#include "Constants.h"
#include <thrust/reduce.h>

DataPoints *AllocateDataPoints(int num_features, int num_data_points, const bool malloc_managed)
{
	if (MEASURE_TIME)
	{
		timer_memory_allocation_gpu->Start();
	}
	// if (malloc_managed == false)
	// {
	// 	DataPoints *point;
	// 	hipMalloc(&point, sizeof(DataPoints));
	// 	cudaCheckError();
	// 	int *tmp1 =(int* ) malloc(sizeof(int));
	// 	*tmp1 = num_data_points;

	// 	hipMemcpy(&(point->num_data_points), tmp1, sizeof(int), hipMemcpyHostToDevice);
	// 	// point->num_data_points = num_data_points;
	// 	cudaCheckError();

	// 	hipMalloc(&(point->cluster_id_of_point), sizeof(int) * num_data_points);
	// 	cudaCheckError();
	// 	hipMemset(point->cluster_id_of_point, 0, sizeof(int) * num_data_points);
	// 	cudaCheckError();

	// 	// point->num_features = num_features;
	// 	int *tmp2 =(int* ) malloc(sizeof(int));
	// 	*tmp2= num_features;
	// 	hipMemcpy(&(point->num_features), tmp2, sizeof(int), hipMemcpyHostToDevice);
	// 	cudaCheckError();

	// 	hipMalloc(&(point->features_array), sizeof(*(point->features_array)) * point->num_features);
	// 	cudaCheckError();

	// 	for (int feature = 0; feature < point->num_features; ++feature)
	// 	{
	// 		hipMalloc(&(point->features_array[feature]), sizeof(MyDataType) * point->num_data_points);
	// 		cudaCheckError();
	// 		hipMemset(point->features_array[feature], 0, sizeof(MyDataType) * point->num_data_points);
	// 		cudaCheckError();
	// 	}
	// 	free(tmp1);
	// 	free(tmp2);
	// 	return point;
	// }
	// else
	// {

	DataPoints *point;
	hipMallocManaged(&point, sizeof(DataPoints));
	cudaCheckError();

	point->num_data_points = num_data_points;
	hipMallocManaged(&(point->cluster_id_of_point), sizeof(int) * num_data_points);
	cudaCheckError();
	hipMemset(point->cluster_id_of_point, 0, sizeof(int) * num_data_points);
	cudaCheckError();

	point->num_features = num_features;
	hipMallocManaged(&(point->features_array), sizeof(*(point->features_array)) * point->num_features);
	cudaCheckError();

	for (int feature = 0; feature < point->num_features; ++feature)
	{
		hipMallocManaged(&(point->features_array[feature]), sizeof(MyDataType) * point->num_data_points);
		cudaCheckError();
		hipMemset(point->features_array[feature], 0, sizeof(MyDataType) * point->num_data_points);
		cudaCheckError();
	}
	if (MEASURE_TIME)
	{
		timer_memory_allocation_gpu->Stop();
		timer_memory_allocation_gpu->Elapsed();
	}
	return point;
	// }
}

void DeallocateDataPoints(DataPoints *data_points)
{
	if (MEASURE_TIME)
	{
		timer_memory_allocation_gpu->Start();
	}
	for (int f = 0; f < data_points->num_features; f++)
	{
		hipFree(data_points->features_array[f]);
	}
	hipFree(data_points->features_array);
	hipFree(data_points->cluster_id_of_point);
	hipFree(data_points);
	if (MEASURE_TIME)
	{
		timer_memory_allocation_gpu->Stop();
		timer_memory_allocation_gpu->Elapsed();
	}
}

MyDataType Distance(const DataPoints *p1, const DataPoints *p2, const int point_id, const int cluster_id)
{
	MyDataType error = 0;
	for (int feature = 0; feature < p2->num_features; ++feature)
	{
		error += (p1->features_array[feature][cluster_id] - p2->features_array[feature][point_id]) * (p1->features_array[feature][cluster_id] - p2->features_array[feature][point_id]);
	}
	return error;
}
#include <unistd.h>
MyDataType MeanSquareError(const DataPoints *point, const DataPoints *centroid)
{
	MyDataType error = 0.0;
	for (int i = 0; i < point->num_data_points; ++i)
	{
		error += Distance(centroid, point, i, point->cluster_id_of_point[i]);
	}
	return error / point->num_data_points;
}

#define INDEX_CLUSTER(c, f, num_clusters) (f * num_clusters + c)

template <int F_NUM>
__global__ void CaleculateErrorsForEachPoint(const DataPoints *points,const DataPoints *centroids, MyDataType *sum_erros,const int num_clusters,const int num_points,const int active_threads_count)
{// acctualy, one thrad calcualtes error for two points
	extern __shared__ MyDataType shm_e1[];
	const int tid = threadIdx.x;
	const int gid_read = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	const int gid_write = blockIdx.x * blockDim.x + threadIdx.x;

	int c1, c2;

	// shm_e1[INDEX_E1(tid)] = 0;
	if (gid_read >= num_points)
	{
		return;
	}
	// if (tid >= active_threads_count)
	// {
	// 	return;
	// }
	c1 = points->cluster_id_of_point[gid_read];
	if (gid_read + blockDim.x < num_points)
	{
		c2 = points->cluster_id_of_point[gid_read + blockDim.x];
	}

	MyDataType error = 0;
	for (int f = 0; f < F_NUM; ++f)
	{
		MyDataType tmp = centroids->features_array[f][c1] - points->features_array[f][gid_read];

		error += tmp * tmp;
		if (gid_read + blockDim.x < num_points)
		{
			tmp = centroids->features_array[f][c2] - points->features_array[f][gid_read + blockDim.x];
			error += tmp * tmp;
		}
	}

	sum_erros[gid_write] = error;
}
template<int F_NUM>
MyDataType MeanSquareErrorParallel(const DataPoints *points, const DataPoints *centroids)
{
	int N = points->num_data_points;
	int num_threads = 1024;
	int num_blocks = std::ceil(N / num_threads / 2);
	int num_clusters=centroids->num_data_points;
	MyDataType *errors;
	hipMallocManaged(&errors, sizeof(MyDataType) * N / 2);
	cudaCheckError();
	// hipMemset(errors, 0, sizeof(MyDataType) * N / 2);
	cudaCheckError();
	size_t shm_e_size = sizeof(MyDataType) * F_NUM * num_clusters;
	int act = num_threads;
	CaleculateErrorsForEachPoint<F_NUM><<<num_blocks, num_threads, shm_e_size>>>(points, centroids, errors, num_clusters, N, act);
	hipDeviceSynchronize();
	cudaCheckError();
	auto res = thrust::reduce(errors, errors + N / 2, 0.0);
	hipDeviceSynchronize();
	cudaCheckError();


	MyDataType err = res / (MyDataType)points->num_data_points;
	hipFree(errors);
	cudaCheckError();
	return err;
}

template MyDataType MeanSquareErrorParallel<NUM_FEATURES>(const DataPoints *points, const DataPoints *centroids);

// DataPoints *ReadCsv()
// {
// 	std::vector<Point> points;
// 	std::string line;
// 	std::ifstream file("/home/jan/Desktop/K-Means-Clustering-CUDA/mall_data.csv");
// 	// std::ifstream file("../mall_data.csv");
// 	while (std::getline(file, line))
// 	{
// 		std::stringstream lineStream(line);
// 		std::string bit;
// 		float x, y;
// 		getline(lineStream, bit, ',');
// 		x = std::stof(bit);
// 		getline(lineStream, bit, '\n');
// 		y = stof(bit);

// 		points.push_back(Point(x, y));
// 	}
// 	file.close();

// 	DataPoints *point = AllocateDataPoints(2, points.size());
// 	int i = 0;
// 	for (std::vector<Point>::iterator it = points.begin(); it != points.end(); ++it)
// 	{
// 		float XY[2];
// 		XY[0] = it->x;
// 		XY[1] = it->y;
// 		for (int feature = 0; feature < point->num_features; ++feature)
// 		{
// 			point->features_array[feature][i] = XY[feature];
// 		}
// 		point->cluster_id_of_point[i] = it->cluster;
// 		i++;
// 	}
// 	return point;
// }

void SaveCsv(const DataPoints *point, const std::string file_name)
{
	std::ofstream myfile;
	std::remove(file_name.c_str());
	myfile.open(file_name);
	myfile << "x,y,z,c" << std::endl;

	for (int i = 0; i < point->num_data_points; ++i)
	{
		for (int feature = 0; feature < point->num_features; ++feature)
		{
			myfile << point->features_array[feature][i];
			myfile << ",";
		}

		myfile << point->cluster_id_of_point[i] << std::endl;
	}

	myfile.close();
}