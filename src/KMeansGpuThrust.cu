#include "hip/hip_runtime.h"
#include "KMeansGpuThrust.h"

#include <iostream>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include "CudaCheckError.h"

#include "FindClosestCentriods.h"
#include "Config.h"
#include "vector"
#include "string"

template <int F_NUM>
MyDataType KMeansOneIterationGpuThurst(DataPoints *points, DataPoints *centroids)
{
	const int N = points->num_data_points;
	const int num_threads = 1024;
	const int num_points = points->num_data_points;
	const int num_features = F_NUM;
	const int num_clusters = centroids->num_data_points;
	int num_blocks = (int)std::max(std::ceil(((double)N / (double)num_threads)), 1.0);
	// get nearest clusters
	const size_t shm_find_closest = sizeof(MyDataType) * num_clusters * NUM_FEATURES + sizeof(MyDataType) * num_threads * NUM_FEATURES;

	FindClosestCentroids<NUM_FEATURES><<<num_blocks, num_threads, shm_find_closest>>>(points->features_array, points->cluster_id_of_point, centroids->features_array, num_points, num_features, num_clusters);

	// get nearest clusters
	hipDeviceSynchronize();
	cudaCheckError();

	// count number of points belonging to each cluster
	int count[centroids->num_data_points];
	for (int c = 0; c < centroids->num_data_points; c++)
	{
		count[c] = thrust::count(points->cluster_id_of_point, points->cluster_id_of_point + num_points, c);
		cudaCheckError();
	}
	// count number of points belonging to each cluster

	// find new positions of the clusters

	int **keys_copy;
	hipMallocManaged(&keys_copy, sizeof(int *) * F_NUM);

	MyDataType **features_copy;
	hipMallocManaged(&features_copy, sizeof(MyDataType *) * F_NUM);

	MyDataType **sumed_position_out;
	hipMallocManaged(&sumed_position_out, sizeof(MyDataType *) * F_NUM);

	int **keys_out;
	hipMallocManaged(&keys_out, sizeof(MyDataType *) * F_NUM);

	for (int f = 0; f < F_NUM; ++f)
	{
		hipMallocManaged(&(keys_copy[f]), sizeof(int) * num_points);
		cudaCheckError();
		hipMemcpyAsync(keys_copy[f], points->cluster_id_of_point, sizeof(int) * num_points, hipMemcpyDeviceToDevice);
		cudaCheckError();

		hipMallocManaged(&(features_copy[f]), sizeof(MyDataType) * num_points);
		cudaCheckError();
		hipMemcpyAsync(features_copy[f], points->features_array[f], sizeof(MyDataType) * num_points, hipMemcpyDeviceToDevice);
		cudaCheckError();

		hipMallocManaged(&(sumed_position_out[f]), sizeof(MyDataType) * num_clusters);
		cudaCheckError();

		hipMallocManaged(&(keys_out[f]), sizeof(int) * num_clusters);
		cudaCheckError();
	}

	hipDeviceSynchronize();
	for (int f = 0; f < num_features; ++f)
	{
		// mozna by jakich prefetch zrobic tych danych tj. wczeniej poleciec async copy i miec odrazy wszyskie dane
		// co jeszce o tym pomyslec
		cudaCheckError();

		thrust::sort_by_key(keys_copy[f], keys_copy[f] + num_points, features_copy[f]);

		auto new_end = thrust::reduce_by_key(keys_copy[f], keys_copy[f] + num_points, features_copy[f], keys_out[f], sumed_position_out[f]);
		cudaCheckError();

		// if (feature + 1 < num_features)
		// {
		// 	hipMemcpyAsync(keys_copy, points->cluster_id_of_point, sizeof(int) * num_points, hipMemcpyDeviceToDevice);
		// 	cudaCheckError();

		// 	hipMemcpyAsync(features_copy, points->features_array[feature + 1], sizeof(MyDataType) * num_points, hipMemcpyDeviceToDevice);
		// 	cudaCheckError();
		// }
	}
	hipDeviceSynchronize();
	for (int f = 0; f < num_features; ++f)
	{

		for (auto c = 0; c < num_clusters; c++)
		{
			centroids->features_array[f][c] = sumed_position_out[f][c] / count[c];
		}
	}
	hipFree(keys_copy);
	cudaCheckError();
	hipFree(features_copy);
	cudaCheckError();
	hipFree(sumed_position_out);
	cudaCheckError();
	hipFree(keys_out);
	cudaCheckError();
	return MeanSquareErrorParallel<F_NUM>(points, centroids);
	// find new positions of the clusters
}

template MyDataType KMeansOneIterationGpuThurst<NUM_FEATURES>(DataPoints *points, DataPoints *centroids);