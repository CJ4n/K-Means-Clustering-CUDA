#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <unistd.h>
#include <iomanip>

#include "Config.h"
#include "CudaCheckError.h"
#include "DataPoints.h"
#include "GeneratePoints.h"
#include "KMeansCpu.h"
#include "KMeansGpu.h"
#include "KMeansGpuThrust.h"
#include "Timer.h"

DataPoints *GetCentroids(DataPoints *point, int num_clusters)
{
	DataPoints *centroids = AllocateDataPoints(point->num_features, num_clusters);

	for (int i = 0; i < num_clusters; ++i)
	{
		// int n = rand() % point->num_data_points;
		for (int feature = 0; feature < point->num_features; ++feature)
		{
			centroids->features_array[feature][i] = point->features_array[feature][i];
		}

		centroids->cluster_id_of_point[i] = i;
	}
	return centroids;
}

double kMeansClustering(DataPoints *point, const int num_clusters, MyDataType (*k_means_one_iteration_algorithm)(DataPoints *, DataPoints *))
{
	DataPoints *centroids = GetCentroids(point, num_clusters);
	MyDataType error = 0;
	MyDataType last_error = 0;
	int epoch = 0;

	while (1)
	{
		error = k_means_one_iteration_algorithm(point, centroids);
		hipDeviceSynchronize();
		cudaCheckError();
		if (!DEBUG_PROGRAM)
		{
			std::cout << "EPOCH: " << epoch << " ERROR: " << error << std::endl;
		}
		if (END_AFTER_N_EPOCHES)
		{
			if (epoch >= NUM_EPOCHES)
			{
				break;
			}
		}
		else
		{
			if (epoch == 0)
			{
				last_error = epoch;
			}
			else
			{
				if (std::abs(last_error - error) < EPS)
				{
					std::cout << "Diff between last error and currnet is closer then " << EPS << ", so ending computation";
					break;
				}
				last_error = error;
			}
		}
		epoch++;
	}
	DeallocateDataPoints(centroids);
	return error;
}

double RunKMeansClustering(MyDataType (*k_means_one_iteration_algorithm)(DataPoints *, DataPoints *), std::string alg_name,
						   const int num_points, const int num_cluster, GpuTimer *timer)
{
	timer_memory_allocation_gpu->total_time = 0;
	std::srand(0);
	DataPoints *point = GeneratePoints(NUM_FEATURES, num_points);

	timer->Start();
	double error = kMeansClustering(point, num_cluster, k_means_one_iteration_algorithm);
	timer->Stop();
	timer->Elapsed();
	DeallocateDataPoints(point);
	return error;
}

int main(int argc, char **argv)
{
	InitTimers();

	std::cout << std::setprecision(10);
	if (!DEBUG_PROGRAM)
	{
		//________________________________THRUST________________________________
		std::cout << "----------------THURST----------------" << std::endl;
		// RunKMeansClustering(KMeansOneIterationGpuThurst, "THRUST",   NUM_POINTS, NUM_CLUSTERS,  timer_thurst_version);
		//________________________________THRUST________________________________

		//__________________________________CPU_________________________________
		std::cout << "-----------------CPU------------------" << std::endl;
		RunKMeansClustering(KMeansOneIterationCpu, "CPU", NUM_POINTS, NUM_CLUSTERS, timer_cpu_version);
		//__________________________________CPU_________________________________

		//__________________________________GPU_________________________________
		std::cout << "-----------------GPU------------------" << std::endl;
		RunKMeansClustering(KMeansOneIterationGpu<NUM_FEATURES>, "GPU", NUM_POINTS, NUM_CLUSTERS, timer_gpu_version);
		//__________________________________GPU_________________________________

		std::cout << "THURST implementation:  " << timer_thurst_version->total_time << "ms" << std::endl;

		std::cout << "CPU implementation:     " << timer_cpu_version->total_time << "ms" << std::endl;
		std::cout << "GPU implementation:     " << timer_gpu_version->total_time << "ms" << std::endl;
		if (MEASURE_TIME)
		{
			std::cout << "compute_centroids:      " << timer_compute_centroids->total_time << "ms" << std::endl;
			std::cout << "find_closest_centroids: " << timer_find_closest_centroids->total_time << "ms" << std::endl;
			std::cout << "timer_memory_allocation_gpu: " << timer_memory_allocation_gpu->total_time << "ms" << std::endl;
			std::cout << "timer_data_generation: " << timer_data_generation->total_time << "ms" << std::endl;
		}
	}
	else // test for many combinations of params
	{
		for (int c = 3; c < 10; c++)
		{
			for (int i = 17; i < 25; i++)
			{
				int num_points = 1 << i;
				const MyDataType exact_error = RunKMeansClustering(KMeansOneIterationCpu, "CPU", num_points, c, timer_cpu_version);
				const MyDataType gpu_error = RunKMeansClustering(KMeansOneIterationGpu<NUM_FEATURES>, "GPU", num_points, c, timer_gpu_version);
				std::cout << "num_cluster: " << c << " num_feature: " << NUM_FEATURES << " num_points: i<<" << i << std::endl;

				if (std::abs(exact_error - gpu_error) > 10e-7)
				{
					std::cout << "<<|||||||||||||||||||||||||dfd|||"
							  << "num_cluster: " << c << " num_feature: " << NUM_FEATURES << " num_points: i<<" << i << "||||||||||||||||||||||||||||" << std::endl;
					std::cout << "exact_error: " << exact_error << std::endl;
					std::cout << "gpu_error:   " << gpu_error << std::endl;
				}
			}
		}
	}

	DeleteTimers();
	return 0;
}
